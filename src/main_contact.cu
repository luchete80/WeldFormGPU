#include "hip/hip_runtime.h"

#include "Domain.h"

#include "cuda/Domain_d.cuh" 
#include "cuda/Mechanical.cu" 

#define TAU		0.005
#define VMAX	10.0

#include <sstream>
#include <fstream> 
#include <iostream>

//#include "cuda/KickDriftSolver.cu"
//#include "cuda/SolverLeapfrog.cu"
#include "cuda/SolverFraser.cu"
#include "cuda/Mesh.cuh"
#include "cuda/Mesh.cu"

bool contact = true;

//#include "Vector.h"

void UserAcc(SPH::Domain_d & domi)
{
		ApplyBCVelKernel	<<<domi.blocksPerGrid,domi.threadsPerBlock >>>(&domi, 2, make_double3(0.,0.,0.));
		hipDeviceSynchronize();
    double vbc;
    if (domi.Time < TAU) vbc = VMAX/TAU*domi.Time;
    else            vbc = VMAX;
    
    if (domi.contact){
      domi.trimesh->SetVel(make_double3(0.,0.,-vbc));
      hipDeviceSynchronize();
    }
    else {
      ApplyBCVelKernel	<<<domi.blocksPerGrid,domi.threadsPerBlock >>>(&domi, 3, make_double3(0.,0.,-vbc));
      hipDeviceSynchronize();
    }
}


void report_gpu_mem()
{
    size_t free, total;
    hipMemGetInfo(&free, &total);
    std::cout << "Free = " << free << " Total = " << total <<std::endl;
}


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
//https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

using std::cout;
using std::endl;

//__host__		SPH::Domain dom;

void WriteCSV(char const * FileKey, double3 *x, double3 *varv, int count){
	std::ostringstream oss;
	std::string fn(FileKey);
	
	oss << "X, Y, Z, dx, dy, dz"<<endl;;
	
	//#pragma omp parallel for schedule(static) num_threads(Nproc)
	// #ifdef __GNUC__
	// for (size_t i=0; i<Particles.Size(); i++)	//Like in Domain::Move
	// #else
	for (int i=0; i<count; i++)//Like in Domain::Move
	//#endif
	{
			oss << x[i].x<<", "<<x[i].y<<", "<<x[i].z<<", "<<varv[i].x<<
", "<<varv[i].y<<
", "<<varv[i].z<<			endl;
		
		//Particles[i]->CalculateEquivalentStress();		//If XML output is active this is calculated twice
		//oss << Particles[i]->Sigma_eq<< ", "<< Particles[i]->pl_strain <<endl;
	}

	std::ofstream of(fn.c_str(), std::ios::out);
	of << oss.str();
	of.close();
}

int main(int argc, char **argv) //try
{
	
	
	cout << "Initializing"<<endl;
	SPH::Domain dom;//Cannot be defined as _device
	// //OR cudamalloc((void**)&correctBool, sizeof(int));
	// hipMallocManaged(&dom, sizeof(SPH::Domain));
	// new(dom) SPH::Domain();
	
	SPH::Domain_d *dom_d;
	report_gpu_mem();
	gpuErrchk(hipMallocManaged(&dom_d, sizeof(SPH::Domain)) );
	report_gpu_mem();

  dom.Dimension	= 3;
  dom.Nproc	= 4;
  //dom.Kernel_Set(Qubic_Spline);

//  dom.Scheme	= 0;
//     	dom.XSPH	= 0.5; //Very important

	double dx,h,rho,K,G;
	double R,L,n;

	R	= 0.15;
	L	= 0.56;
	n	= 30.0;		//in length, radius is same distance

	rho	= 2700.0;
	K	= 6.7549e10;
	G	= 2.5902e10;
	
	//dx = 0.030; //THIS IS FOR TESTING Original 6,5mm, 8mm 10mm, 12,5 and 15mm
  dx = 0.015; //THIS IS FOR TESTING Original 6,5mm, 8mm 10mm, 12,5 and 15mm
	h	= dx*1.2; //Very important

	double Cs	= sqrt(K/rho);

  double timestep = (0.4*h/(Cs));

	cout<<"deltat  = "<<timestep<<endl;
	cout<<"Cs = "<<Cs<<endl;
	// cout<<"K  = "<<K<<endl;
	// cout<<"G  = "<<G<<endl;
	// cout<<"Fy = "<<Fy<<endl;
	
	// dom.GeneralAfter = & UserAcc;
	dom.DomMax(0) = L;
	dom.DomMin(0) = -L;
  dom_d->GeneralAfter = & UserAcc;
	cout << "Creating Domain"<<endl;
  //NEVER USE 1 IN CONTACT PROBLEMS (IS CONTACT SURFACE ID)
	dom.AddCylinderLength(0, Vector(0.,0.,-L/20.), R, L + 2.*L/20.,  dx/2., rho, h, false); 
	cout << "Particle count:" <<dom.Particles.size()<<endl;
  
  /************************* NEW CONTACT THING *////////////////////////////////////////////
  	double cyl_zmax = dom.Particles[dom.Particles.size()-1]->x(2) + 0.99999995 * dom.Particles[dom.Particles.size()-1]->h ;

  SPH::TriMesh mesh;
	mesh.AxisPlaneMesh(2,false,Vector(-0.3,-0.3, cyl_zmax),Vector(0.3,0.3, cyl_zmax),30);
	//cout << "Plane z" << *mesh.node[0]<<endl;
  //mesh.CalcSpheres(); //DONE ONCE
  double hfac = 1.1;
  dom_d->first_fem_particle_idx = dom.Particles.size(); // TODO: THIS SHOULD BE DONE AUTOMATICALLY
  int solid_count = dom.Particles.size(); //BEFORE ADDING CONTACT MESH
  
  dom.AddTrimeshParticles(mesh, hfac, 11); //TO SHARE SAME PARTICLE NUMBER
  dom_d->contact_surf_id = 11; //TO DO: AUTO! From Domain_d->AddTriMesh
  
  //TODO: Mesh has to be deleted
  SPH::TriMesh_d *mesh_d;
  gpuErrchk(hipMallocManaged(&mesh_d, sizeof(SPH::TriMesh_d)) );
  mesh_d->AxisPlaneMesh(2,false,make_double3(-0.3,-0.3, cyl_zmax),make_double3(0.3,0.3, cyl_zmax),30);
  
  cout << "Domain Size "<<dom.Particles.size()<<endl;
	//BEFORE ALLOCATING 
  int particlecount = dom.Particles.size();
  // //cout << "Particles "<<
	dom_d->SetDimension(particlecount);	 //AFTER CREATING DOMAIN
  dom_d->solid_part_count = solid_count;  //AFTER SET DIMENSION
  dom_d->trimesh = mesh_d; //TODO: CHECK WHY ADDRESS IS LOST
  if (dom_d->trimesh ==NULL)
    cout << "ERROR. No mesh defined"<<endl;
  
  
  /********************************** END NEW CONTACT THING */////////////////////////////////
	
  //SPH::Domain	dom;
	//double3 *x =  (double3 *)malloc(dom.Particles.size());
	double3 *x =  new double3 [dom.Particles.size()];
	for (int i=0;i<dom.Particles.size();i++){
		//cout <<"i; "<<i<<endl;
		//x[i] = make_double3(dom.Particles[i]->x);
		x[i] = make_double3(double(dom.Particles[i]->x(0)), double(dom.Particles[i]->x(1)), double(dom.Particles[i]->x(2)));
	}
	int size = dom.Particles.size() * sizeof(double3);
	cout << "Copying to device..."<<endl;
	hipMemcpy(dom_d->x, x, size, hipMemcpyHostToDevice);


	for (int i=0;i<dom.Particles.size();i++){
		x[i] = make_double3(0.,0.,0.);
	}
	hipMemcpy(dom_d->v, x, size, hipMemcpyHostToDevice);
  
	cout << "copied"<<endl;
  
	
	cout << "Setting values"<<endl;
	dom_d->SetDensity(rho);
	dom_d->Set_h(h);
	cout << "done."<<endl;

	double *m =  new double [dom.Particles.size()];
  double totmass = 0.;
	for (size_t a=0; a<dom.Particles.size(); a++){
		m[a] = dom.Particles[a]->Mass;
    totmass +=m[a];
  }
	hipMemcpy(dom_d->m, m, dom.Particles.size() * sizeof(double), hipMemcpyHostToDevice);	
  hipMemcpy(&dom_d->totmass, &totmass, sizeof(double),hipMemcpyHostToDevice);
		
		// // std::cout << "Particle Number: "<< dom.Particles.size() << endl;
     	// // double x;

	//MODIFY
	double *T 			=  new double [dom.Particles.size()];
	int 	*BC_type 	=  new int 		[dom.Particles.size()];
	int bcpart = 0;
	for (size_t a=0; a<dom.Particles.size(); a++){
		double xx = dom.Particles[a]->x(0);
		BC_type[a]=0;
		T[a] = 20.;
		if ( xx < -L/2.0 ) {
			bcpart++;
			BC_type[a]=1;
		}
	}		
	cout << "BC particles"<<bcpart<<endl;
	hipMemcpy(dom_d->T, T, dom.Particles.size() * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dom_d->BC_T, BC_type, dom.Particles.size() * sizeof(int), hipMemcpyHostToDevice);
	
	dom_d->Alpha = 0.0;//For all particles		
	dom_d->SetShearModulus(G);	// 
  

  bool *not_write = new bool[dom_d->first_fem_particle_idx];
  for (int i=0;i< dom_d->first_fem_particle_idx;i++){
    not_write[i] = false;
  }
  cout << "Defining surface "<<endl;
	for (size_t a=0; a< dom_d->first_fem_particle_idx; a++) {
    dom.Particles[a]->Cs			= Cs;  //THIS IS CRUCIAL
		double z = dom.Particles[a]->x(2);
		if ( z < 0 ){
			dom.Particles[a]->ID=2;	
      not_write[a] = true;
		}
    if (!contact)
      if ( z > L ){
        dom.Particles[a]->ID=3;
        not_write[a] = true;  
      }
	}
  
  hipMemcpy(dom_d->not_write_surf_ID, not_write, dom_d->first_fem_particle_idx * sizeof(bool), hipMemcpyHostToDevice);
	
	dom_d->SetFreePart(dom); //All set to IsFree = true in this example
	dom_d->SetID(dom); 
	dom_d->SetCs(dom);
	
	dom_d->SetSigmay(300.e6);
	

        // // timestep = (0.3*h*h*rho*dom.Particles[0]->cp_T/dom.Particles[0]->k_T);	
		// // cout << "Time Step: "<<timestep<<endl;
		// // //timestep=1.e-6;
		// // //0.3 rho cp h^2/k
	
		
	// dom.WriteCSV("maz");
	
	// WriteCSV_kernel<<<1,1>>>(&dom);

	cout << "Solving "<<endl;
	//CheckData<<<1,1>>>(dom_d);
	//hipDeviceSynchronize(); //Crashes if not Sync!!!
	
	

	
	cout << "Time Step: "<<dom_d->deltat<<endl;
	WriteCSV("test_inicial.csv", x, dom_d->u_h, dom.Particles.size());

	dom_d->auto_ts = false;
  if (contact)
    dom_d->contact = true;
  dom_d->Alpha = 0.7;
  
  dom_d->friction_sta =   dom_d->friction_dyn = 0.;
	//dom_d->MechSolve(0.0101,1.0e-4);
  
  //New solver
  timestep = (dom_d->Alpha*h/(Cs+VMAX));
  dom_d->deltat = timestep;
  //dom_d->MechKickDriftSolve(0.0101,1.0e-4);
  //LEAPFROG IS WORKING WITH ALPHA = 1
  //KICKDRIFT IS NOT 

  //dom_d->MechFraserSolve(0.0101,1.0e-4);
  dom_d->MechFraserSolve(3.0*timestep,timestep);
  //FOR DEBUG PURPOSES
  //dom_d->MechFraserSolve(4*timestep,timestep);
  
  //First example
  // dom_d->deltat = 1.0e-7;
	// dom_d->auto_ts = false;
  // dom_d->Alpha = 1.0;
	//dom_d->MechSolve(0.00101,1.0e-4);


	hipMemcpy(T, dom_d->T, sizeof(double) * dom.Particles.size(), hipMemcpyDeviceToHost);	
	
        // return 0;
	//WriteCSV("test.csv", x, dom_d->u_h, dom.Particles.size());
	dom_d->WriteCSV("test.csv");
	
	hipFree(dom_d);
	//report_gpu_mem();
	cout << "Program ended."<<endl;
}
//MECHSYS_CATCH
