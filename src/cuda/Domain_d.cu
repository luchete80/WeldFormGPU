#include "hip/hip_runtime.h"
#include "Domain_d.cuh"
#include "Functions.cuh"
#include "Domain.h"

#include <chrono>
//#include <time.h>       /* time_t, struct tm, difftime, time, mktime */
#include <ctime> //Clock

#include "Mesh.h"
#include "cudautils.cuh"
#include "Mesh.cuh"
#include "lsdynaReader.h"
//Else (offset)
//Allocating from host

using namespace LS_Dyna;

namespace SPH {
void Domain_d::SetDimension(const int &particle_count){
	this->particle_count = particle_count;
	//Allocae arrays (as Structure of arryays, SOA)

	hipMalloc((void **)&x, particle_count * sizeof (double3));
	
	hipMalloc((void **)&h, 	particle_count * sizeof (double));
	hipMalloc((void **)&m, 	particle_count * sizeof (double));
	hipMalloc((void **)&rho, particle_count * sizeof (double));
	
	///////////
	//THERMAL //
  if (thermal_solver){
    hipMalloc((void **)&k_T, 	particle_count * sizeof (double));
    hipMalloc((void **)&cp_T, 	particle_count * sizeof (double));
      
    hipMalloc((void **)&T		, particle_count * sizeof (double));
    hipMalloc((void **)&Ta		, particle_count * sizeof (double));
    hipMalloc((void **)&Tb		, particle_count * sizeof (double));
    
    hipMalloc((void **)&BC_T, particle_count * sizeof (int));


    // hipMalloc((void **)&T_inf , particle_count * sizeof (double));
    // hipMalloc((void **)&q_conv, particle_count * sizeof (double));
    // hipMalloc((void **)&h_conv, particle_count * sizeof (double));	
  }  
	//Host things
	T_h =  new double  [particle_count];
	x_h =  new double3 [particle_count];
	v_h =  new double3 [particle_count];
	u_h =  new double3 [particle_count];
	a_h =  new double3 [particle_count];
  
  h_h =  new double [particle_count];
  
  
  normal_h = new double3 [particle_count];
  nb_h = new unsigned int [particle_count];
	
  if (realloc_ID) {delete [] ID_h;}
  ID_h =  new int    [particle_count];
	
	sigma_eq_h =  new double [particle_count];
	pl_strain_h = new double [particle_count];
	
	p_h =  new double [particle_count];
	rho_h =  new double [particle_count];
	
	max_deltat_h = new double [particle_count];
	
	hipMalloc((void **)&dTdt	, particle_count * sizeof (double));
	//printf("Size of dTdt: %d, particle count %d\n",sizeof(dTdt)/sizeof (double),particle_count);
  
  //Contact 
  contneib_count_h = new int [particle_count];

	//Nb data
	hipMalloc((void **)&neib_offs	, (particle_count + 1) * sizeof (int));
	hipMalloc((void **)&neib_part	, (particle_count * 100) * sizeof (int));
	
	hipMalloc((void **)&partdata, sizeof(PartData_d));
  
  hipMalloc((void **)&test, sizeof(particle_count)* sizeof (double));
	
	////////////////////////////
	/////// SPH ////////////////
	hipMalloc((void **)&SumKernel, 	particle_count  * sizeof (double));	
	
	////////////////////////////
	//// MECHANICAL DATA ///////
	////////////////////////////
	hipMalloc((void **)&v, particle_count * sizeof (double3));
	hipMalloc((void **)&a, particle_count * sizeof (double3));
	hipMalloc((void **)&u, particle_count * sizeof (double3));
	
	hipMalloc((void **)&va, particle_count * sizeof (double3));
	hipMalloc((void **)&vb, particle_count * sizeof (double3));
	
	hipMalloc((void **)&p, 			particle_count * sizeof (double));	
  
  report_gpu_mem_();
  
	/// DensitySolid ///
	//DensitySolid (PresEq[i], Cs[i], P0[i],p[j], rho_0[i]);
  cout << "Allocating  state vars "<<endl;
	hipMalloc((void **)&PresEq, 	particle_count  * sizeof (double));	
	hipMalloc((void **)&Cs, 			particle_count  * sizeof (double));		
	hipMalloc((void **)&P0, 			particle_count  * sizeof (double));		
	hipMalloc((void **)&FPMassC, particle_count  * sizeof (double));	
	hipMalloc((void **)&rho_0, 	particle_count  * sizeof (double));	

	hipMalloc((void **)&G, 	particle_count  * sizeof (double));	
	
	// hipMalloc((void **)&rhoa, 	particle_count  * sizeof (double));	
	// hipMalloc((void **)&rhob, 	particle_count  * sizeof (double));	
	hipMalloc((void **)&drho, 	particle_count  * sizeof (double));		
	
	// STRESS AND STRAIN TENSORS - FLATTENED ARRAY!!!!
	hipMalloc((void **)&sigma		, particle_count  * 6 * sizeof (double));		
	hipMalloc((void **)&strrate	, particle_count  * 6 * sizeof (double));			
	hipMalloc((void **)&rotrate	, particle_count  * 6 * sizeof (double));		 //ANTISYMM
  
	
	hipMalloc((void **)&shearstress	, particle_count  * 6 * sizeof (double));		 //ANTISYMM
	// hipMalloc((void **)&shearstressa	, particle_count  * 6 * sizeof (double));		 //ANTISYMM
	// hipMalloc((void **)&shearstressb	, particle_count  * 6 * sizeof (double));		 //ANTISYMM

	hipMalloc((void **)&sigma_eq		, particle_count  * sizeof (double));		
	hipMalloc((void **)&pl_strain	, particle_count  * sizeof (double));	
	hipMalloc((void **)&sigma_y		, particle_count  * sizeof (double));		
	
	hipMalloc((void **)&strain		, particle_count  * 6 * sizeof (double));		
	// hipMalloc((void **)&straina	, particle_count  * 6 * sizeof (double));		
	// hipMalloc((void **)&strainb	, particle_count  * 6 * sizeof (double));		
  
  hipMalloc((void **)&eff_strain_rate	, particle_count  * sizeof (double));		
	
	// BOUNDARY CONDITIONS
	hipMalloc((void **)&IsFree	, particle_count  * sizeof (bool));	
	hipMalloc((void **)&NoSlip	, particle_count  * sizeof (bool));
	hipMalloc((void **)&NSv, 		particle_count  * sizeof (double3));	
	hipMalloc((void **)&ID, 			particle_count  * sizeof (int));	
	
  
  

  hipMalloc((void**)&mat, particle_count * sizeof(Material_ *));

	//////////////////////////
	/////// TENSILE INST /////
	hipMalloc((void **)&TI, 					particle_count  * sizeof (double));	
	hipMalloc((void **)&TIn, 				particle_count  * sizeof (double));		
	hipMalloc((void **)&TIInitDist, 	particle_count  * sizeof (double));		
	hipMalloc((void **)&TIR, 				6 * particle_count  * sizeof (double));	

	hipMalloc((void **)&max_deltat, 	particle_count  * sizeof (double));		
  
  
  /////////////CONTACT////////////////
	hipMalloc((void **)&normal,          particle_count * sizeof (double3));	
  
  hipMalloc((void **)&contneib_count,  particle_count * sizeof (int));
  hipMalloc((void **)&contneib_part,   particle_count * 20 * sizeof (int));
  hipMalloc((void **)&contneib_offs,   particle_count * sizeof (int));
  hipMalloc((void **)&contforce,       particle_count * sizeof (double3));
  
  hipMalloc((void **)&mesh_id,         particle_count * sizeof (int));
  
  contforce_h = new double3 [particle_count];
  
  //////////////////////////
  ///// ENERGY /////////////
  hipMalloc((void **)&int_energy,   particle_count * sizeof (double));
  hipMalloc((void **)&kin_energy,   particle_count * sizeof (double));
	
	//////////////////////////
	/// CORRECTIONS /////////
	hipMalloc((void **)&VXSPH, 	particle_count  * sizeof (double3));		
  
  
  //pplane_h =  new double [particle_count-solid_part_count]; ////TEST

  
  //trimesh = NULL;
	//hipMalloc((void **)&partdata->dTdt,particle_count * sizeof (double)); //TODO, pass to PartData
	
	// hipMalloc((void**)&ppArray_a, 10 * sizeof(int*));
	// for(int i=0; i<10; i++) {
		// hipMalloc(&someHostArray[i], 100*sizeof(int)); /* Replace 100 with the dimension that u want */
	// }
	
	Alpha = 1.;
  Beta  = 0.;
	auto_ts = true;
	
	deltat	= 0.0;
	deltatint	= 0.0;
	deltatmin	= 0.0;
	sqrt_h_a = 0.0025;	
	
	//Initiate pl_strain, is it necessary
	double *k_ =  new double[particle_count];
	for (int i=0;i<particle_count;i++){
		k_[i] = 0.;
	}
	int size = particle_count * sizeof(double);
	hipMemcpy(this->pl_strain, k_, size, hipMemcpyHostToDevice);
	delete k_;
  
  //Change to first particle
  hipMalloc((void **)&not_write_surf_ID, 	particle_count  * sizeof (bool));		
  bool *surf = new bool[particle_count];
  for (int i=0;i<particle_count;i++){
    surf[i] = false;
  }
  hipMemcpy(not_write_surf_ID, surf,  particle_count * sizeof(bool), hipMemcpyHostToDevice);


  
  //SetDouble(this->int_energy,0.);
	double *ki_ =  new double[particle_count];
	for (int i=0;i<particle_count;i++){
		ki_[i] = 0.0;
	}

	hipMemcpy(int_energy, ki_, particle_count * sizeof(double), hipMemcpyHostToDevice);
	delete ki_;	
  
  solid_part_count = particle_count;
	
	//To allocate Neighbours, it is best to use a equal sized double array in order to be allocated once
  
  cout << "Domain allocation done. "<<endl;
  report_gpu_mem_();
}

void __host__ Domain_d::ReadFromLSdyna(const char *fName){
  lsdynaReader reader(fName);
  
  particle_count = reader.m_elem_count_type[_SPH_];
  cout << "Particles readed: "<< reader.m_elem_count_type[_SPH_]<<endl;
  //SetDimension(particle_count);

  isdim_reserved = true;
  x_h =  new double3 [particle_count];
  m_h =  new double [particle_count];
  
  for (int i=0;i<reader.m_elem.size();i++) {
    if (reader.m_elem[i].m_type == _SPH_){
      LS_Dyna::ls_node n = reader.getElemNode(i,0);
      //cout << "Node XYZ"<< n.m_x[0]<< ", "<<n.m_x[1]<< ", "<<n.m_x[2]<< ", "<<endl;
      x_h[i] = make_double3(double(n.m_x[0]), double(n.m_x[1]), double(n.m_x[2]));
      m_h[i] = reader.m_elem[i].mass;
    }
  }
  cout << "Reading "<<reader.m_set_nod.size()<< " sets."<<endl;
  if (reader.m_set_nod.size()>0) {
    realloc_ID = true;
    this->ID_h = new int [particle_count];
    cout << "Assigning "<<reader.m_set_nod.size()<<" IDs"<<endl;
    for (int p=0;p<particle_count;p++) {ID_h[p] = 0;}
    for (int s=0;s<reader.m_set_nod.size();s++){
      cout << "Set "<< s<< ", Reading "<<reader.m_set_nod[s].node.size()<< " nodes."<<endl; 
      for (int n=0;n<reader.m_set_nod[s].node.size();n++){      
        //cout << "Node "<<n << ", pos "<<reader.m_set_nod[s].node[n]<<endl;
		int elpos = reader.m_node[reader.m_set_nod[s].node[n]].id_sph_el;
		//cout << "elpos: "<<elpos<<endl;
        // if (reader.m_set_nod[s].node[n]>= particle_count){
          // cout << "ERROR. Node "<<n << ", pos "<<reader.m_set_nod[s].node[n]<<endl;
        // } else {
        ID_h[elpos] = reader.m_set_nod[s].id;          
        //}
        //ID_h[n]=0;
      }
    }
  }
  //delete ID_h;

  
  cout << "Done. "<<endl;
  
}

void ReadFromLSdyna(const char *fName, Domain_d *dom) {
  // lsdynaReader reader(fName);
  // particle_count = reader.m_spc_nod.size();
  // cout << "Particles readed: "<< reader.m_elem_count_type[_SPH_]<<endl;
  // SetDimension(particle_count);

	// double3 *x =  new double3 [particle_count];
	// for (int i=0;i<particle_count;i++){

		// //x[i] = make_double3(double(dom.Particles[i]->x(0)), double(dom.Particles[i]->x(1)), double(dom.Particles[i]->x(2)));
	// }
	// int size = particle_count * sizeof(double3);
	// cout << "Copying to device "<<particle_count<< " particle properties ..."<<endl;
	// //hipMemcpy(dom_d->x, x, size, hipMemcpyHostToDevice);

  
  // for (int i=0;i<reader.m_elem.size();i++) {
    // if (reader.m_elem[i].m_type == _SPH_){
      // LS_Dyna::ls_node n = reader.getElemNode(i,0);
      // //cout << "Node XYZ"<< n.m_x[0]<< ", "<<n.m_x[1]<< ", "<<n.m_x[2]<< ", "<<endl;
    // }
  // }  
}

__host__ void Domain_d::SetFreePart(const Domain &dom){
	bool *k_ =  new bool[particle_count];
	for (int i=0;i<particle_count;i++){
		k_[i] = dom.Particles[i]->IsFree;
	}
	int size = particle_count * sizeof(bool);
	hipMemcpy(this->IsFree, k_, size, hipMemcpyHostToDevice);
	delete k_;	
}

__host__ void Domain_d::SetID(const Domain &dom){
	int *k_ =  new int[particle_count];
	for (int i=0;i<particle_count;i++){
		k_[i] = dom.Particles[i]->ID;
    if (dom.Particles[i]->ID<0) printf("ERROR, NEGATIVE ID \n");
    this->ID_h[i] = dom.Particles[i]->ID;
    //cout << "ID: "<<dom.Particles[i]->ID;
	}
  cout << "Copying "<<particle_count<<" particles id"<<endl;
	int size = particle_count * sizeof(int);
	hipMemcpy(this->ID, k_, size, hipMemcpyHostToDevice);
	delete k_;	
}

__host__ void Domain_d::SetCs(const Domain &dom){
	double *k_ =  new double[particle_count];
	for (int i=0;i<particle_count;i++){
		k_[i] = dom.Particles[i]->Cs;
	}
	int size = particle_count * sizeof(double);
	hipMemcpy(this->Cs, k_, size, hipMemcpyHostToDevice);
	delete k_;	
}

//TODO: MOVE TO TEMPLATE
__host__ void Domain_d::SetDouble(double *arr, double val){
	int *k_ =  new int[particle_count];
	for (int i=0;i<particle_count;i++){
		k_[i] = val;
	}
	int size = particle_count * sizeof(double);
	hipMemcpy(arr, k_, size, hipMemcpyHostToDevice);
	delete k_;	
}

// template <typename T>
// __host__ void Domain_d::SetType(T* arr, double val){
	// T *k_ =  new T[particle_count];
	// for (int i=0;i<particle_count;i++){
		// k_[i] = val;
	// }
	// int size = particle_count * sizeof(T);
	// hipMemcpy(arr, k_, size, hipMemcpyHostToDevice);
	// delete k_;	
// }

// template __host__ void Domain_d::SetType<double>(double *arr, double val);

void Domain_d::CheckData(){
	printf("dTdt partdta: %d",sizeof(this->dTdt)/sizeof(double));
	printf("dTdt[200] %f",dTdt[200]);
	printf("neibpart %f",neib_part[300000]);
	//dom->CheckData();
}

__global__ void CheckData(Domain_d *dom){
	//printf("dTdt partdta: %d",sizeof(dom->partdata->dTdt)/sizeof(double));
	dom->CheckData();
}

__device__ void Domain_d::AssignMatAddress(int i){
  if (i< solid_part_count){
    mat[i] = &materials[0];
  
    //printf("part %d mat E%f\n", i, materials[0].Elastic().E());
  }
}

__global__ void AssignMatAddressKernel(Domain_d *dom){
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  dom->AssignMatAddress(i);
}

///////// TODO: Chage to pointer directly instead of int
__device__ void Domain_d::AssignTrimeshID(int i, int id, m/*, int start, int end*/){
  if (i> first_fem_particle_idx[m] && i < first_fem_particle_idx[m]+trimesh[m]->elemcount)
    mesh_id[i] = id;
}

__global__ void AssignTrimeshIDKernel(Domain_d *dom, int id, int m/*, int start, int end*/){
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  dom->AssignTrimeshID(i, id, m/*start, end*/);
}

__device__ void Domain_d::AssignTrimeshAddress(int id, TriMesh_d *mesh){
  this->trimesh[id]=mesh;
  printf( "Added mesh node count %d\n",trimesh[0]->nodecount);
}

__global__ void AssignTrimeshAddressKernel(Domain_d *dom, int id, TriMesh_d *mesh){ //Assign particle data to material array to zero arra

  dom->AssignTrimeshAddress(id, mesh);
}

void Domain_d::Set_h(const double &k){
	double *k_ =  new double[particle_count];
	for (int i=0;i<particle_count;i++){
		k_[i] = k;
	}
	int size = particle_count * sizeof(double);
	hipMemcpy(this->h, k_, size, hipMemcpyHostToDevice);
	h_glob = k;
	delete k_;
}

void Domain_d::SetConductivity(const double &k){
	double *k_ =  new double[particle_count];
	for (int i=0;i<particle_count;i++){
		k_[i] = k;
	}
	int size = particle_count * sizeof(double);
	hipMemcpy(this->k_T, k_, size, hipMemcpyHostToDevice);
	delete k_;
}

void Domain_d::SetSigmay(const double &k){
	double *k_ =  new double[particle_count];
	for (int i=0;i<particle_count;i++){
		k_[i] = k;
	}
	int size = particle_count * sizeof(double);
	hipMemcpy(this->sigma_y, k_, size, hipMemcpyHostToDevice);
	delete k_;
}

void Domain_d::SetShearModulus(const double &k){
	double *k_ =  new double[particle_count];
	for (int i=0;i<particle_count;i++){
		k_[i] = k;
	}
	int size = particle_count * sizeof(double);
	hipMemcpy(this->G, k_, size, hipMemcpyHostToDevice);
	delete k_;
}


void Domain_d::SetDensity(const double &k){
	double *k_ =  new double[particle_count];
	for (int i=0;i<particle_count;i++){
		k_[i] = k;
	}
	int size = particle_count * sizeof(double);
	hipMemcpy(this->rho, k_, size, hipMemcpyHostToDevice);
	hipMemcpy(this->rho_0, k_, size, hipMemcpyHostToDevice);
	delete k_;
}

void Domain_d::SetHeatCap(const double &cp){
	double *cp_ =  new double[particle_count];
	for (int i=0;i<particle_count;i++){
		cp_[i] = cp;
	}
	int size = particle_count * sizeof(double);
	hipMemcpy(this->cp_T, cp_, size, hipMemcpyHostToDevice);
	delete cp_;
}

// // Templatize data type, and host and device vars (of this type)
// template <typename T> copydata (const Domain &d, T *var_h, T *var_d){
	// T *var_h =  (Vector *)malloc(dom.Particles.size());
	// for (int i=0;i<dom.Particles.size();i++){
		// var_h[i] = dom.Particles[i]->T;
	// }
	// int size = dom.Particles.size() * sizeof(Vector);
	// hipMemcpy(this->T, T, size, hipMemcpyHostToDevice);
// }

//TEMPORARY, UNTIL EVERYTHING WILL BE CREATED ON DEVICE
void __host__ Domain_d::CopyData(const Domain& dom){
	
	//TODO TEMPLATIZE THIS!!
	double *T =  (double *)malloc(dom.Particles.size());
	for (int i=0;i<dom.Particles.size();i++){
		T[i] = dom.Particles[i]->T;
	}
	int size = dom.Particles.size() * sizeof(double);
	hipMemcpy(this->T, T, size, hipMemcpyHostToDevice);

	// for (int i=0;i<dom.Particles.size();i++){
		// T[i] = dom.Particles[i]->cp_T;
	// }
	// int size = dom.Particles.size() * sizeof(double);
	// hipMemcpy(this->cp_T, T, size, hipMemcpyHostToDevice);
	
}

void __device__ Domain_d::CalcThermalTimeStep(){
	deltat = 0.3*h[0]*h[0]*rho[0]*cp_T[0]/k_T[0];
	printf("Time Step: %f\n",deltat);
}


//NEXT SOLVER
// void Domain_d::ThermalSolve(const double &tf){
	
	
// }

Domain_d::~Domain_d(){
	
		hipFree(a);		
		hipFree(v);

		hipFree(h);		
		hipFree(m);
		hipFree(rho);

		hipFree(neib_offs);
		hipFree(neib_part);		
}

    // // Create host pointer to array-like storage of device pointers
    // Obj** h_d_obj = (Obj**)malloc(sizeof(Obj*) * 3); //    <--------- SEE QUESTION 1
    // for (int i = 0; i < 3; i++) {
        // // Allocate space for an Obj and assign
        // hipMalloc((void**)&h_d_obj[i], sizeof(Obj));
        // // Copy the object to the device (only has single scalar field to keep it simple)
        // hipMemcpy(h_d_obj[i], &(h_obj[i]), sizeof(Obj), hipMemcpyHostToDevice);
    // }

    // /**************************************************/
    // /* CREATE DEVICE ARRAY TO PASS POINTERS TO KERNEL */
    // /**************************************************/

    // // Create a pointer which will point to device memory
    // Obj** d_d_obj = NULL;
    // // Allocate space for 3 pointers on device at above location
    // hipMalloc((void**)&d_d_obj, sizeof(Obj*) * 3);
    // // Copy the pointers from the host memory to the device array
    // hipMemcpy(d_d_obj, h_d_obj, sizeof(Obj*) * 3, hipMemcpyHostToDevice);


#include <cstdio>

void Domain_d::WriteCSV(char const * FileKey){
	FILE *f = fopen(FileKey,"w");;
	
	fprintf(f, "ID, X, Y, Z, posX,posY,posZ,Ux, Uy, Uz, Vx, Vy, Vz, Ax, Ay, Az, rho, p, SigmaEq, Pl_Strain, Nb, ContNb, CFx, CFy, CFz, Nx, Ny, Nz, h\n");

	// for (size_t i=0; i<Particles.Size(); i++)	//Like in Domain::Move
  double ppl;
	for (int i=0; i<particle_count; i++) {
		fprintf(f,"%d, %.6e, %.6e, %.6e, %.6e, %.6e, %.6e, %.6e, %.6e, %.6e, %.6e, %.6e, %.6e, %.6e, %.6e, %.6e, %.6e, %.6e, %.6e, %.6e, %d, %d, %.6e, %.6e, %.6e, %.6e, %.6e, %.6e, %6e\n", 
							ID_h[i],
              x_h[i].x,x_h[i].y,x_h[i].z, 
              x_h[i].x,x_h[i].y,x_h[i].z, 
              u_h[i].x,u_h[i].y,u_h[i].z,
							v_h[i].x,v_h[i].y,v_h[i].z, 
							a_h[i].x,a_h[i].y,a_h[i].z,
							//ID[i],
						rho_h[i],
						p_h[i],
						sigma_eq_h[i],
						pl_strain_h[i],
            nb_h[i],
            contneib_count_h[i],
            contforce_h[i].x,contforce_h[i].y,contforce_h[i].z,
            normal_h[i].x,normal_h[i].y,normal_h[i].z,
            h_h[i]
            );
		//Particles[i]->CalculateEquivalentStress();		//If XML output is active this is calculated twice
		//oss << Particles[i]->Sigma_eq<< ", "<< Particles[i]->pl_strain <<endl;
	}

 fclose(f);
}

__global__ void CalcMinTimeStepKernel(Domain_d *dom){
	
	dom->CalcMinTimeStep();	//Stablish deltatmin based on acceleration
	
}

//TODO: CHANGE TO MECH
__device__ void Domain_d::CalcMinTimeStep(){
		int i = threadIdx.x + blockDim.x*blockIdx.x;
		//THIS WAS IN LASTCOMPUTEACCELERATION original code
		// //Min time step check based on the acceleration
		if (i<particle_count){
			double test	= 0.0;
			deltatmin	= deltatint;
			double sqrt_h_a = 0.0025;
			//Appears to be safe
			//https://stackoverflow.com/questions/8416374/several-threads-writing-the-same-value-in-the-same-global-memory-location
	
			//ORIGINAL: test = sqrt(h[i]/length(a[i]));
			if (IsFree[i]) {
				
				//test = h[i]/(Cs[i]+length(v[i]));
				//max_deltat[i]=0.6*test;
				//if (deltatmin > (0.6*test)) 
					//deltatmin = 0.6*test;
        
				test = sqrt(h[i]/length(a[i]));
				max_deltat[i]=sqrt_h_a*test;				
				if (deltatmin > (sqrt_h_a*test)) 
						deltatmin = sqrt_h_a*test;


						
						//printf("particle i: %d Min time step %f\n",i,deltatmin);
				
			}
		}		
}

__host__ void Domain_d::AdaptiveTimeStep(){
		if (deltatint>deltatmin) {
		if (deltat<deltatmin)
			deltat		= 2.0*deltat*deltatmin/(deltat+deltatmin);
		else
			deltat		= deltatmin;
	} else {
		if (deltatint!=deltat)
			deltat		= 2.0*deltat*deltatint/(deltat+deltatint);
		else
			deltat		= deltatint;
	}
}

///// NOT USED ///////
//If this is called, all particle has to be reallocated
__device__ void Domain_d::AddTrimeshParticles(TriMesh_d* mesh, double hfac, int id){

	first_fem_particle_idx[trimesh_count] = particle_count;
	printf("First Rigid particle of mesh %d:",first_fem_particle_idx[trimesh_count]);
	double Density =0.;
	double h;
	bool Fixed = false;	//Always are fixed ...
	contact_surf_id[trimesh_count] = id;
  
	trimesh[trimesh_count] = mesh;
  trimesh_count++;
  
	// for ( int e = 0; e < mesh.element.size(); e++ ){
		// Vector pos = mesh.element[e]->centroid;
		// double h = hfac * mesh.element[e]->radius;
		// Particles.push_back(new Particle(id,pos,Vector(0,0,0),0.0,Density,h,Fixed));
		// // Particles[first_fem_particle_idx + e] -> normal  = mesh.element[e] -> normal;
		// // Particles[first_fem_particle_idx + e] -> element = e; 
	// }
}

__device__ void Domain_d::ApplyAxiSymmBC(int bc_1, int bc_2){ //Apply to all particles or only to BCs. If Not all (!=-1), 
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  if (i<particle_count){
    double abs_a = sqrt (a[i].x*a[i].x + a[i].y * a[i].y);
    if (abs_a>1.0e-3){
      if (x[i].x > h[i]/2.0 && a[i].x > 1.0e-2){
      //if ID==2 
        //if (ID[i]==2 || ID[i]==3){
          double alpha = atan(x[i].y/x[i].x); //TODO: SAVE IT AT THE BEGINING
          double beta_a  = atan(a[i].y/a[i].x);
          printf("Part %d, ax, %.6e ay %.6e\n", i, a[i].x,a[i].y);
          printf("alpha %.6e , beta %.6e \n", alpha, beta_a);
          double mod = abs_a * cos(beta_a - alpha);
          a[i].x = mod * cos(alpha);
          a[i].y = mod * sin(alpha);
          //if (abs(a[i].x)>1.0e-3 && abs )
          printf( "corrected acc axy %.6e %.6e\n",a[i].x ,a[i].y);
        //} else if (ID[i] == 4){
          a[i].x = a[i].y = 0.0;
        //}
      
      }
    }//abs a <
  }
}



__global__ void ApplyAxiSymmBCKernel(Domain_d *dom){
  dom->ApplyAxiSymmBC();
}

__global__ void AddTrimeshParticlesKernel(Domain_d *dom, TriMesh_d* mesh, double hfac, int id){
  dom->AddTrimeshParticles(mesh, hfac, id);
}

// THIS SHOULD BE DONE
	// if (deltatint>deltatmin)
	// {
		// if (deltat<deltatmin)
			// deltat		= 2.0*deltat*deltatmin/(deltat+deltatmin);
		// else
			// deltat		= deltatmin;
	// }
	// else
	// {
		// if (deltatint!=deltat)
			// deltat		= 2.0*deltat*deltatint/(deltat+deltatint);
		// else
			// deltat		= deltatint;
	// }
	
	// if (contact){
		// if (min_force_ts < deltat)
		// //cout << "Step size changed minimum Contact Forcess time: " << 	min_force_ts<<endl;
		// deltat = min_force_ts;
	// }

	// if (deltat<(deltatint/1.0e5))
		// //cout << "WARNING: Too small time step, please choose a smaller time step initially to make the simulation more stable"<<endl;
		// throw new Fatal("Too small time step, please choose a smaller time step initially to make the simulation more stable");
// }


};//SPH
