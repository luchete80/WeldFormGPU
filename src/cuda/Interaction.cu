#include "hip/hip_runtime.h"
#include "Domain_d.cuh"
#include "Functions.cuh"
#include "tensor.cuh"

namespace SPH {
// THIS CAUSED UNRESOLVED EXTERNAL, WITH INLINE TENSOR FUNCTIONS 
__global__ void CalcForcesMember(PartData_d *partdata){
	
	partdata->CalcForce2233(0,0.0);
}

__global__ void CalcForcesKernel(Domain_d *dom_d){
	//int i = threadIdx.x + blockDim.x*blockIdx.x;
	dom_d->CalcForce2233(0,0.0);
}

#define NEIBS(i, k) partdata->neib_part[partdata->neib_offs[i]+k]
__global__ void CalcForce2233(PartData_d *partdata){
	
	//partdata->CalcForce2233(0,0.0);
	CalcForcesExt(partdata);
}

// __device__ tensor3 Identity(){
	// tensor3 ret;
	// ret(0,0) = ret(1,1) = ret(2,2) = 1.;
	// //ret[1][1]=ret[2][2]=1.;
	
	// return ret;
// }

// __device__ tensor3::tensor3(){

// }


//THIS IS NOT A FUNCTION MEMBER
// THIS ALLOWS FOR COMPILING VIA NON SEPARABLE COMPILATION!!!!
__device__ void CalcForcesExt(PartData_d *partdata){
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	int Dimension = 3; //TODO, put in another 
	int neibcount;
	#ifdef FIXED_NBSIZE
	neibcount = partdata->neib_offs[i];
	#else
	neibcount =	partdata->neib_offs[i+1] - partdata->neib_offs[i];
	#endif
	printf("Solving\n");
	for (int k=0;k < neibcount; k++) { //Or size
		int j = NEIBS(i,k);
		//double h	= partdata->h[i]+P2->h)/2;
		double3 xij = partdata->x[i] - partdata->x[j];
		double rij = length(xij);
		double di=0.0,dj=0.0,mi=0.0,mj=0.0;
	
		if (!partdata->IsFree[i]) {
			di = DensitySolid(partdata->PresEq[i], partdata->Cs[j], partdata->P0[j],partdata->p[i], partdata->rho_0[j]);
			mi = partdata->FPMassC[i] * partdata->m[j];
		} else {
			di = partdata->rho[i];
			mi = partdata->m[i];
		}
		if (!partdata->IsFree[j]) {
			dj = DensitySolid (partdata->PresEq[i], partdata->Cs[i], partdata->P0[i],partdata->p[j], partdata->rho_0[i]);
			mj = partdata->FPMassC[j] * partdata->m[i];
		} else {
			dj = partdata->rho[j];
			mj = partdata->m[j];
		}	
		
		double3 vij	= partdata->v[i] - partdata->v[j];
		double h_ = (partdata->h[i] + partdata->h[j])/2.0;
			
		//double GK	= GradKernel(Dimension, KernelType, rij/h, h);
		double GK	= GradKernel(3, 0, rij/h_, h_);
		double K	= Kernel(3, 0, rij/h_, h_);

		////// Artificial Viscosity
		tensor3 PIij;
		//set_to_zero(PIij);
		if (partdata->Alpha!=0.0 || partdata->Beta!=0.0)
		{
			double MUij = h_*dot(vij,xij)/(rij*rij+0.01*h_*h_);					///<(2.75) Li, Liu Book
			double Cij;
			double Ci,Cj;
			if (!partdata->IsFree[i]) Ci = SoundSpeed(partdata->PresEq[j], partdata->Cs[j], di, partdata->rho_0[j]); 
			else 											Ci = SoundSpeed(partdata->PresEq[i], partdata->Cs[i], di, partdata->rho_0[i]);
			if (!partdata->IsFree[j]) Cj = SoundSpeed(partdata->PresEq[j], partdata->Cs[i], dj, partdata->rho_0[i]); 
			else 											Cj = SoundSpeed(partdata->PresEq[j], partdata->Cs[j], dj, partdata->rho_0[j]);
			Cij = 0.5*(Ci+Cj);
			
			if (dot(vij,xij)<0) PIij = (partdata->Alpha*Cij*MUij + partdata->Beta*MUij*MUij)/(0.5*(di+dj)) * Identity();		///<(2.74) Li, Liu Book
		}
		
	}//neibcount	
}

//TODO; COMPARE WITH ORIGINAL IN PARTDATA
//This exludes thermal, 
//__global__ void CalcForce2233(PartMassVolInfo
																//PartMechData *pmd
																//){
	
//Be a part data member???
//CALLED BY GLOBAL
//TODO; DIVIDE PARTDATA INTO DIFFERENT FIELDS
__device__ /*inline*/ void PartData_d::CalcForce2233(
	/* const double &Dimension*/
	int KernelType,
	float XSPH)
{
}



__device__ /*inline*/ void Domain_d::CalcForce2233(
	/* const double &Dimension*/
	int KernelType,
	float XSPH)
{
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	
	if ( i < particle_count ) {
	int Dimension = 3; //TODO, put in another 
	int neibcount;
	#ifdef FIXED_NBSIZE
	neibcount = neib_offs[i];
	#else
	neibcount =	neib_offs[i+1] - neib_offs[i];
	#endif
	//printf("Solving\n");
	tensor3 StrainRate,RotationRate;
	
	for (int k=0;k < neibcount; k++) { //Or size
		//if fixed size i = part * NB + k
		//int j = neib[i][k];
		int j = NEIB(i,k);
		//double h	= partdata->h[i]+P2->h)/2;
		double3 xij = x[i] - x[j];
		double rij = length(xij);
		double di=0.0,dj=0.0,mi=0.0,mj=0.0;
		
		//Artifficial visc
		// double Alpha	= (P1->Alpha + P2->Alpha)/2.0;
		// double Beta	= (P1->Beta + P2->Beta)/2.0;
		
		if (!IsFree[i]) {
			di = DensitySolid(PresEq[i], Cs[j], P0[j],p[i], rho_0[j]);
			mi = FPMassC[i] * m[j];
		} else {
			di = rho[i];
			mi = m[i];
		}
		if (!IsFree[j]) {
			dj = DensitySolid (PresEq[i], Cs[i], P0[i],p[j], rho_0[i]);
			mj = FPMassC[j] * m[i];
		} else {
			dj = rho[j];
			mj = m[j];
		}

		double3 vij	= v[i] - v[j];
		double h_ = (h[i] + h[j])/2.0;
			
		//double GK	= GradKernel(Dimension, KernelType, rij/h, h);
		double GK	= GradKernel(3, KernelType, rij/h_, h_);
		double K	= Kernel(3, 0, rij/h_, h_);
		
		////// Artificial Viscosity
		tensor3 PIij;
		//set_to_zero(PIij);

		if (Alpha!=0.0 || Beta!=0.0)
		{
			double MUij = h_*dot(vij,xij)/(rij*rij+0.01*h_*h_);					///<(2.75) Li, Liu Book
			double Cij;
			double Ci,Cj;
			if (!IsFree[i]) Ci = SoundSpeed(PresEq[j], Cs[j], di, rho_0[j]); else Ci = SoundSpeed(PresEq[i], Cs[i], di, rho_0[i]);
			if (!IsFree[j]) Cj = SoundSpeed(PresEq[j], Cs[i], dj, rho_0[i]); else Cj = SoundSpeed(PresEq[j], Cs[j], dj, rho_0[j]);
			Cij = 0.5*(Ci+Cj);
			
			//printf("C %f %f\n",Ci,Cj);
			if (dot(vij,xij)<0) 
				PIij = (Alpha*Cij*MUij+Beta*MUij*MUij)/(0.5*(di+dj)) * Identity();		///<(2.74) Li, Liu Book
		}
		
		//printf("i %d, Ti %f\n",i, T[i]);
		
		tensor3 Sigma,Sigmaj,Sigmai;
		// set_to_zero(Sigmaj);
		// set_to_zero(Sigmai);
		
		//TODO: CONVERT FLATTENED ARRAY TO TENSOR
		//TODO: Avoid temp array conversion and test
		double tempi[6],tempj[6];
		for (int k=0;k<6;k++){ //First the diagonal
			tempi[k]=sigma[6*i+k];
			tempj[k]=sigma[6*j+k];
		}
		
		Sigmai.FromFlatSym(tempi);
		Sigmaj.FromFlatSym(tempj);
		//Sigmai = Sigma[i];
		//Sigmaj = Sigma[j];

		//THIS IS COMMENTED IN THE ORIGINAL CODE
//		if (IsFree[i]) Sigmai = P1->Sigma; else  Sigmai = P2->Sigma;
//		if (IsFree[j]) Sigmaj = P2->Sigma; else  Sigmaj = P1->Sigma;
		
		// Tensile Instability //////////////////////
		tensor3 TIij;
		tensor3 TIRi, TIRj;
		// for (int k=0;k<6;k++) {
			// TIRi.FromFlatSymPtr(TIR[6*i+k]);
			// TIRj.FromFlatSymPtr(TIR[6*j+k]);
		// }
		//TODO: CONVERT TIR FROM FLATTENED ARRAY TO TENSOR
		//set_to_zero(TIij);
		if (TI[i] > 0.0 || TI[j] > 0.0) 
			TIij = pow((K/Kernel(Dimension, KernelType, (TIInitDist[i] + TIInitDist[j])/(2.0*h_), h_)),(TIn[i] + TIn[j])/2.0)*(TIRi+TIRj);
			//TIij = pow((K/m_kernel.W((P1->TIInitDist + P2->TIInitDist)/(2.0*h))),(P1->TIn+P2->TIn)/2.0)*(P1->TIR+P2->TIR); //COMMENTED IN ORIGINAL CODE
		
		// NoSlip BC velocity correction 		////////////////////////////////
		double3 vab = make_double3(0.0);
		if (IsFree[i]*IsFree[j]) {
			vab = vij;
		} else {
			if (NoSlip[i] || NoSlip[j] ) {
				// No-Slip velocity correction
				if (IsFree[i])	vab = v[i] - (2.0f*v[j]- NSv[j]); 
				else vab = (2.0f*v[i]- NSv[i]) - v[j];
			}
			// Please check
			if (!(NoSlip[i] || NoSlip[j])) {
				if (IsFree[i]) vab = v[i] - v[j]; else vab = v[i] - v[j];
//				if (IsFree[i]) vab.x = v[i](0) + v[j]b(0); else vab.x = -v[i]b(0) - v[j](0);
			}
		} //Are not both fixed
		
		// set_to_zero(StrainRate);
		// set_to_zero(RotationRate);

		////////////////////////////////////
		// // Calculation strain rate tensor
		////////////////////////////////////
		if (i==1250 || j==1250)
			printf("Time, i,j,vab: %.4e %d %d %f %f %f\n",Time, i,j,vab.x,vab.y,vab.z);
		StrainRate(0,0) = 2.0*vab.x*xij.x;
		StrainRate(0,1) = vab.x*xij.y+vab.y*xij.x;
		StrainRate(0,2) = vab.x*xij.z+vab.z*xij.x;
		StrainRate(1,0) = StrainRate(0,1);
		StrainRate(1,1) = 2.0*vab.y*xij.y;
		StrainRate(1,2) = vab.y*xij.z+vab.z*xij.y;
		StrainRate(2,0) = StrainRate(0,2);
		StrainRate(2,1) = StrainRate(1,2);
		StrainRate(2,2) = 2.0*vab.z*xij.z;
		StrainRate	= -0.5 * GK * StrainRate;
		
		if (i==1250 || j==1250)
			printf("Strain Rate i %f %f %f\n",StrainRate(0,0),StrainRate(1,1),StrainRate(2,2));

		// // Calculation rotation rate tensor
		RotationRate(0,1) = vab.x*xij.y-vab.y*xij.x;
		RotationRate(0,2) = vab.x*xij.z-vab.z*xij.x;
		RotationRate(1,2) = vab.y*xij.z-vab.z*xij.y;
		RotationRate(1,0) = -RotationRate(0,1);
		RotationRate(2,0) = -RotationRate(0,2);
		RotationRate(2,1) = -RotationRate(1,2);
		RotationRate	  = -0.5 * GK * RotationRate;
		
		//printf("Particle %d strain rate: %f %f %f\n",i,StrainRate(0,0),StrainRate(1,1),StrainRate(2,2));

		// XSPH Monaghan
		if (XSPH != 0.0  && (IsFree[i]*IsFree[j])) {
			//omp_set_lock(&P1->my_lock);
			VXSPH[i] += XSPH*mj/(0.5f*(di+dj))*K*(-vij);
			//omp_unset_lock(&P1->my_lock);
	
		
			//NOT WRITE IN THE OTHER PART!
			//omp_set_lock(&P2->my_lock);
			// VXSPH[j] += XSPH*mi/
			// (0.5*(di+dj))*
			// K*vij;
			//omp_unset_lock(&P2->my_lock);
		}		
		
		
		double3 temp = make_double3(0.0);
		double temp1 = 0.0;
		
		//if (GradientType == 0)
		if (i == 1250)
			printf("Particle 1250 Time %.4e, Sigmaizz %f , Sigmajzz %f\n",Time, Sigmai(2,2),Sigmaj(2,2));
		temp = ( 1.0/(di*di)*Sigmai + 1.0/(dj*dj)*Sigmaj + PIij /*+ TIij */) * (GK*xij);
			//Mult( GK*xij , ( 1.0/(di*di)*Sigmai + 1.0/(dj*dj)*Sigmaj /*+ PIij + TIij */) , temp); //TODO: TIR AND ARTIFF VISC
		// else
			// Mult( GK*xij , ( 1.0/(di*dj)*(Sigmai + Sigmaj)           + PIij + TIij ) , temp);

		//if (Dimension == 2) temp(2) = 0.0;
		temp1 = dot( vij , GK*xij );

		// Locking the particle 1 for updating the properties
		a[i] 		+= mj * temp;
		drho[i]	+= mj * (di/dj) * temp1;

		if (IsFree[i]) {
			float mj_dj= mj/dj;
			//P1->ZWab	+= mj_dj* K;
			//printf("mj /dj %f\n",mj_dj);
			StrainRate = StrainRate + mj_dj * StrainRate;
			RotationRate = RotationRate + mj_dj * RotationRate;
			
			//P1->RotationRate = P1->RotationRate + mj_dj*RotationRate;
		}
		// else
			// P1->ZWab	= 1.0;

		// if (P1->Shepard)
			// if (P1->ShepardCounter == P1->ShepardStep)
				// P1->SumDen += mj*    K;

	
		// THIS IS THE ORIGINAL
		// Locking the particle 2 for updating the properties
		//omp_set_lock(&P2->my_lock);
			// P2->a		-= mi * temp;
			// P2->dDensity	+= mi * (dj/di) * temp1;
			// if (IsFree[j]) {
				// float mi_di = mi/di;
				// //P2->ZWab	+= mi_di* K;
				// //StrainRate[j]	  += StrainRate[j] + mi_di*StrainRate;
				// //RotationRate[j] += P2->RotationRate + mi_di*RotationRate;

			// }
			// else
				// P2->ZWab	= 1.0;

			// if (P2->Shepard)
				// if (P2->ShepardCounter == P2->ShepardStep)
					// P2->SumDen += mi*    K;

	
		
		}//neibcount

		///// OUTPUT TO Flatten arrays
		RotationRate.ToFlatSymPtr(rotrate,6*i);
		StrainRate.ToFlatSymPtr(strrate,6*i);	//Is the same for antisymm, stores upper diagonal
		if (i==1250)
			printf("TOTAL (SUM) Strain Rate part %d %f %f %f\n",i, StrainRate(0,0),StrainRate(1,1),StrainRate(2,2));
	}//i < partcount
}

}; //SPH