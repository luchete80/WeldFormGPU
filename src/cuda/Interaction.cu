#include "hip/hip_runtime.h"
#include "Domain_d.cuh"
#include "Functions.cuh"
#include "tensor.cuh"

namespace SPH {
__global__ void CalcForce2233(PartData_d *partdata){
	
	partdata->CalcForce2233(0,0.0);
}

//TODO; COMPARE WITH ORIGINAL IN PARTDATA
//This exludes thermal, 
//__global__ void CalcForce2233(PartMassVolInfo
																//PartMechData *pmd
																//){
	
//Be a part data member???
//CALLED BY GLOBAL
//TODO; DIVIDE PARTDATA INTO DIFFERENT FIELDS
__device__ inline void PartData_d::CalcForce2233(
	/* const double &Dimension*/
	const int & KernelType,
	const float &XSPH)
{
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	
	int Dimension = 3; //TODO, put in another 
	int neibcount;
	#ifdef FIXED_NBSIZE
	neibcount = neib_offs[i];
	#else
	neibcount =	neib_offs[i+1] - neib_offs[i];
	#endif
	printf("Solving\n");
	for (int k=0;k < neibcount; k++) { //Or size
		//if fixed size i = part * NB + k
		//int j = neib[i][k];
		int j = NEIB(i,k);
		//double h	= partdata->h[i]+P2->h)/2;
		double3 xij = x[i] - x[j];
		double rij = length(xij);
		double di=0.0,dj=0.0,mi=0.0,mj=0.0;
		
		//Artifficial visc
		// double Alpha	= (P1->Alpha + P2->Alpha)/2.0;
		// double Beta	= (P1->Beta + P2->Beta)/2.0;
		
		if (!IsFree[i]) {
			di = DensitySolid(PresEq[i], Cs[j], P0[j],p[i], rho_0[j]);
			mi = FPMassC[i] * m[j];
		} else {
			di = rho[i];
			mi = m[i];
		}
		if (!IsFree[j]) {
			dj = DensitySolid (PresEq[i], Cs[i], P0[i],p[j], rho_0[i]);
			mj = FPMassC[j] * m[i];
		} else {
			dj = rho[j];
			mj = m[j];
		}

		double3 vij	= v[i] - v[j];
		double h_ = (h[i] + h[j])/2.0;
			
		//double GK	= GradKernel(Dimension, KernelType, rij/h, h);
		double GK	= GradKernel(3, KernelType, rij/h_, h_);
		double K	= Kernel(3, 0, rij/h_, h_);
		
		////// Artificial Viscosity
		tensor3 PIij;
		//set_to_zero(PIij);
		if (Alpha!=0.0 || Beta!=0.0)
		{
			double MUij = h_*dot(vij,xij)/(rij*rij+0.01*h_*h_);					///<(2.75) Li, Liu Book
			double Cij;
			double Ci,Cj;
			if (!IsFree[i]) Ci = SoundSpeed(PresEq[j], Cs[j], di, rho_0[j]); else Ci = SoundSpeed(PresEq[i], Cs[i], di, rho_0[i]);
			if (!IsFree[j]) Cj = SoundSpeed(PresEq[j], Cs[i], dj, rho_0[i]); else Cj = SoundSpeed(PresEq[j], Cs[j], dj, rho_0[j]);
			Cij = 0.5*(Ci+Cj);
			
			if (dot(vij,xij)<0) PIij = (Alpha*Cij*MUij+Beta*MUij*MUij)/(0.5*(di+dj)) * Identity();		///<(2.74) Li, Liu Book
		}
		
		
		tensor3 Sigma,Sigmaj,Sigmai;
		// set_to_zero(Sigmaj);
		// set_to_zero(Sigmai);
		
		//TODO: CONVERT FLATTENED ARRAY TO TENSOR
		Sigmai = Sigma[i];
		Sigmaj = Sigma[j];

		//THIS IS COMMENTED IN THE ORIGINAL CODE
//		if (IsFree[i]) Sigmai = P1->Sigma; else  Sigmai = P2->Sigma;
//		if (IsFree[j]) Sigmaj = P2->Sigma; else  Sigmaj = P1->Sigma;
		
		// Tensile Instability //////////////////////
		tensor3 TIij;
		tensor3 TIRi, TIRj;
		//TODO: CONVERT TIR FROM FLATTENED ARRAY TO TENSOR
		//set_to_zero(TIij);
		if (TI[i] > 0.0 || TI[j] > 0.0) 
			TIij = pow((K/Kernel(Dimension, KernelType, (TIInitDist[i] + TIInitDist[j])/(2.0*h_), h_)),(TIn[i] + TIn[j])/2.0)*(TIRi+TIRj);
			//TIij = pow((K/m_kernel.W((P1->TIInitDist + P2->TIInitDist)/(2.0*h))),(P1->TIn+P2->TIn)/2.0)*(P1->TIR+P2->TIR); //COMMENTED IN ORIGINAL CODE
		
		// NoSlip BC velocity correction 		////////////////////////////////
		double3 vab = make_double3(0.0);
		if (IsFree[i]*IsFree[j]) {
			vab = vij;
		} else {
			if (NoSlip[i] || NoSlip[j] ) {
				// No-Slip velocity correction
				if (IsFree[i])	vab = v[i] - 
				(2.0f*v[j]- NSv[j]); 
				else vab = (2.0f*v[i]- NSv[i]) - v[j];
			}
			// Please check
			if (!(NoSlip[i] || NoSlip[j])) {
				if (IsFree[i]) vab = v[i] - v[j]; else vab = v[i] - v[j];
//				if (IsFree[i]) vab.x = v[i](0) + v[j]b(0); else vab.x = -v[i]b(0) - v[j](0);
			}
		} //Are not both fixed
		
		tensor3 StrainRate,RotationRate;
		// set_to_zero(StrainRate);
		// set_to_zero(RotationRate);

		////////////////////////////////////
		// // Calculation strain rate tensor
		////////////////////////////////////
		StrainRate(0,0) = 2.0*vab.x*xij.x;
		StrainRate(0,1) = vab.x*xij.y+vab.y*xij.x;
		StrainRate(0,2) = vab.x*xij.z+vab.z*xij.x;
		StrainRate(1,0) = StrainRate(0,1);
		StrainRate(1,1) = 2.0*vab.y*xij.y;
		StrainRate(1,2) = vab.y*xij.z+vab.z*xij.y;
		StrainRate(2,0) = StrainRate(0,2);
		StrainRate(2,1) = StrainRate(1,2);
		StrainRate(2,2) = 2.0*vab.z*xij.z;
		StrainRate	= -0.5 * GK * StrainRate;

		// // Calculation rotation rate tensor
		RotationRate(0,1) = vab.x*xij.y-vab.y*xij.x;
		RotationRate(0,2) = vab.x*xij.z-vab.z*xij.x;
		RotationRate(1,2) = vab.y*xij.z-vab.z*xij.y;
		RotationRate(1,0) = -RotationRate(0,1);
		RotationRate(2,0) = -RotationRate(0,2);
		RotationRate(2,1) = -RotationRate(1,2);
		RotationRate	  = -0.5 * GK * RotationRate;

		// XSPH Monaghan
		if (XSPH != 0.0  && (IsFree[i]*IsFree[j])) {
			//omp_set_lock(&P1->my_lock);
			VXSPH[i] += XSPH*mj/(0.5f*(di+dj))*K*(-vij);
			//omp_unset_lock(&P1->my_lock);
	
		
			//NOT WRITE IN THE OTHER PART!
			//omp_set_lock(&P2->my_lock);
			// VXSPH[j] += XSPH*mi/
			// (0.5*(di+dj))*
			// K*vij;
			//omp_unset_lock(&P2->my_lock);
		}		
		
	}//neibcount
	
}

}; //SPH