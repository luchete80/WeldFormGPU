#include "hip/hip_runtime.h"
#include "Domain_d.cuh"
#include "Functions.cuh"
#include <iostream>

#include <chrono>
//#include <time.h>       /* time_t, struct tm, difftime, time, mktime */
#include <ctime> //Clock
#include "tensor3.cu" //INLINE
#include "Interaction.cu"

#include "Geometry.cu"
#include "Contact.cu"

#include "cuNSearch.h"
//For Writing file

//This is temporary since can be used a delta_pl_strain for each particle
#define MIN_PS_FOR_NBSEARCH		1.e-6//TODO: MOVE TO CLASS MEMBER
#include "Mesh.cuh"

using namespace std;

namespace SPH{

__global__ void WholeVelocityKernel(Domain_d *dom_d){
	dom_d->WholeVelocity();
}
//Called by __global__
void __device__ Domain_d::WholeVelocity() {
		int i = threadIdx.x + blockDim.x*blockIdx.x;
	
	if ( i < particle_count ) {
	}
}

//////////////////////////////////////////////////
///// THIS IS ONLY WHEN THERE ARE FIXED PARTICLES!
//////////////////////////////////////////////////
__device__ void Domain_d::PrimaryComputeAcceleration (/*int i*/) {
	int i = threadIdx.x + blockDim.x*blockIdx.x;
		
	if ( i < particle_count ) {
		double3 xij;
		double h_,K;
	int Dimension = 3;
	
	// Summing the smoothed pressure, velocity and stress for fixed particles from neighbour particles
	//Same Material pairs
	int neibcount;
	#ifdef FIXED_NBSIZE
	neibcount = neib_offs[i];
	#else
	neibcount =	neib_offs[i+1] - neib_offs[i];
	#endif
	// printf("neibcount %d\n",neibcount);
	// printf("Nb indexed,i:%d\n",i);
	for (int k=0;k < neibcount;k++) { //Or size
		// P1	= FSMPairs[k][a].first;
		// P2	= FSMPairs[k][a].second;
		int j = NEIB(i,k); //TODO; MAKE A FIXED PAIR
		xij	= x[i]-x[j];
		h_	= (h[i]+ h[j])/2.0;
		double nxij = length(xij);
		//Periodic_X_Correction(xij, h, Particles[P1], Particles[P2]);
		//(size_t const & Dim, size_t const & KT, double const & q, double const & h);
		K	= Kernel(Dimension, 0, nxij/h_, h_);
		if ( !IsFree[i] ) {
				SumKernel[i] += K;
				p[i]	+= p[j] * K /*+ dot(Gravity,xij)*rho[j]*K*/;
				sigma[i] 	 = sigma[i] + K * sigma[j];
				if (NoSlip[i])		NSv[i] 	+= v[j] * K;
		} else {
				SumKernel[j] += K;
				p[j]	+= p[i] * K /*+ dot(Gravity,xij)*rho[i]*K*/;
				sigma[j]	 = sigma[j] + K * sigma[i];
				if (NoSlip[j])		NSv[j] 	+= v[i] * K;

		}	
	}//FIXED neibcount k

	////////////////////////////////////////////////////////////////
	// // Calculateing the finala value of the smoothed pressure, velocity and stress for fixed particles
	////////////////////////////////////////////////////////////////
	
	// #pragma omp parallel for schedule (static) num_threads(Nproc)
	// #ifdef __GNUC__
	// for (size_t i=0; i<FixedParticles.Size(); i++)
	// #else
	// for (int i=0; i<FixedParticles.Size(); i++)
	// #endif
		// if (Particles[FixedParticles[i]]-> ID != contact_surf_id)  //ADDED TO Prevent adding surface (rigid contact) particles
		// if (Particles[FixedParticles[i]]->SumKernel!= 0.0) {
			// size_t a = FixedParticles[i];
			// Particles[a]->Pressure	= Particles[a]->Pressure/Particles[a]->SumKernel;
			// Particles[a]->Sigma	= 1.0/Particles[a]->SumKernel*Particles[a]->Sigma;
			// if (Particles[a]->NoSlip)	Particles[a]->NSv	= Particles[a]->NSv/Particles[a]->SumKernel;

			// // Tensile Instability for fixed soil and solid particles
			// if (Particles[a]->TI > 0.0)
			// {
				// // XY plane must be used, It is very slow in 3D
				// if (Dimension == 2) {
					// double teta, Sigmaxx, Sigmayy, C, S;
					// if ((Particles[a]->Sigma(0,0)-Particles[a]->Sigma(1,1))!=0.0)
						// teta = 0.5*atan(2.0*Particles[a]->Sigma(0,1)/(Particles[a]->Sigma(0,0)-Particles[a]->Sigma(1,1)));
					// else
						// teta = M_PI/4.0;

					// C = cos(teta);
					// S = sin(teta);
					// Sigmaxx = C*C*Particles[a]->Sigma(0,0) + 2.0*C*S*Particles[a]->Sigma(0,1) + S*S*Particles[a]->Sigma(1,1);
					// Sigmayy = S*S*Particles[a]->Sigma(0,0) - 2.0*C*S*Particles[a]->Sigma(0,1) + C*C*Particles[a]->Sigma(1,1);
					// if (Sigmaxx>0) Sigmaxx = -Particles[a]->TI * Sigmaxx/(Particles[a]->Density*Particles[a]->Density); else Sigmaxx = 0.0;
					// if (Sigmayy>0) Sigmayy = -Particles[a]->TI * Sigmayy/(Particles[a]->Density*Particles[a]->Density); else Sigmayy = 0.0;
					// Particles[a]->TIR(0,0) = C*C*Sigmaxx + S*S*Sigmayy;
					// Particles[a]->TIR(1,1) = S*S*Sigmaxx + C*C*Sigmayy;
					// Particles[a]->TIR(0,1) = Particles[a]->TIR(1,0) = S*C*(Sigmaxx-Sigmayy);
				// }
				// else {
					// Mat3_t Vec,Val,VecT,temp;
					// Rotation(Particles[a]->Sigma,Vec,VecT,Val);
					// double pc_ti_inv_d2=Particles[a]->TI/(Particles[a]->Density*Particles[a]->Density);//Precompute some values
					// // if (Val(0,0)>0) Val(0,0) = -Particles[a]->TI * Val(0,0)/(Particles[a]->Density*Particles[a]->Density); else Val(0,0) = 0.0;
					// // if (Val(1,1)>0) Val(1,1) = -Particles[a]->TI * Val(1,1)/(Particles[a]->Density*Particles[a]->Density); else Val(1,1) = 0.0;
					// // if (Val(2,2)>0) Val(2,2) = -Particles[a]->TI * Val(2,2)/(Particles[a]->Density*Particles[a]->Density); else Val(2,2) = 0.0;
					// if (Val(0,0)>0) Val(0,0) = -pc_ti_inv_d2 * Val(0,0); else Val(0,0) = 0.0;
					// if (Val(1,1)>0) Val(1,1) = -pc_ti_inv_d2 * Val(1,1); else Val(1,1) = 0.0;
					// if (Val(2,2)>0) Val(2,2) = -pc_ti_inv_d2 * Val(2,2); else Val(2,2) = 0.0;

					// Mult(Vec,Val,temp);
					// Mult(temp,VecT,Particles[a]->TIR);
				// }
			// }
		// }

	}//i<part_count
}

//IS NOT NECESSARY TO PASS ENTIRE DOMAIN!
void __global__ MoveKernelExt(double3 *v, double3 *va, double3 *vb,
													double *rho, double *rhoa, double *rhob,double *drho,
													double3 *x, double3 *a,
													double3 *u, /*Mat3_t I, */double dt,
													bool FirstStep, int particle_count)
{
	int i = threadIdx.x + blockDim.x*blockIdx.x;
		
	if ( i < particle_count ) {
	if (FirstStep) {
		//printf("First Step\n");
		rhoa[i] = rho[i] - dt/2.0*drho[i];
		va[i] = v[i] - dt/2.0*a[i];
		//printf("First Step!\n");
	}
	rhob[i] = rhoa[i];
	rhoa[i] += dt*drho[i];
	rho[i] = (rhoa[i]+rhob[i])/2.0;
	// if (i==1250){
		// printf("Move. particle 1250 rho %f rhoa %f rhob %f\n",rho[i],rhoa[i],rhob[i]);
	// }
	vb[i] = 	va[i];
	va[i] += dt*a[i];
	v[i] = (va[i] + vb[i])/2.0;
	x[i] += dt*va[i];
	
	u[i] += dt*va[i];

    // Mat2Leapfrog(dt);
	// if (FirstStep) FirstStep = false;
	}
}

// __device__ PartData_d::PrimaryComputeAcceleration(){
	// // Summing the smoothed pressure, velocity and stress for fixed particles from neighbour particles
	
	
	// // Calculateing the finala value of the smoothed pressure, velocity and stress for fixed particles
// }

// void __global__ MechSolveKernel (double dt, PartData_d *partdata) {
	// int i = threadIdx.x+blockDim.x*blockIdx.x;
	// dTdt[i] = 0.;
	
	// int neibcount;
	// #ifdef FIXED_NBSIZE
	// neibcount = neib_offs[i];
	// #else
	// neibcount =	neib_offs[i+1] - neib_offs[i];
	// #endif
	// printf("Solving\n");
	// for (int k=0;k < neibcount;k++) { //Or size
		// //if fixed size i = part * NB + k
		// //int j = neib[i][k];
		// int j = NEIB(i,k);
		// printf("i,j\n",i,j);
		// double3 xij; 
		// xij = x[i] - x[j];
		// double h_ = (h[i] + h[j])/2.0;
		// double nxij = length(xij);
		
		// double GK	= GradKernel(3, 0, nxij/h_, h_);
		// //		Particles[i]->dTdt = 1./(Particles[i]->Density * Particles[i]->cp_T ) * ( temp[i] + Particles[i]->q_conv + Particles[i]->q_source);	
		// //   mc[i]=mj/dj * 4. * ( P1->k_T * P2->k_T) / (P1->k_T + P2->k_T) * ( P1->T - P2->T) * dot( xij , v )/ (norm(xij)*norm(xij));
		// dTdt[i] += m[j]/rho[j]*( 4.0*k_T[i]*k_T[j]/(k_T[i]+k_T[j]) * (T[i] - T[j])) * dot( xij , GK*xij )/(nxij*nxij);
	// }
	// dTdt[i] *=1/(rho[i]*cp[i]);
// }

__device__ __forceinline__ void Domain_d::LastComputeAcceleration(){
	//partdata->();
}

__global__ void PressureKernelExt(double *p, double *PresEq, double *Cs, double *P0,double *Density, double *RefDensity, int particle_count){
	
	int i = threadIdx.x + blockDim.x*blockIdx.x;	
	// if (i == 1250){
		// printf("PresEq[i], Cs[i], P0[i],Density[i], RefDensity[i]: %f %f %f %f %f \n",PresEq[i], Cs[i], P0[i],Density[i], RefDensity[i]);
	// }
	if ( i < particle_count ) {	
		p[i] = EOS(PresEq[i], Cs[i], P0[i],Density[i], RefDensity[i]); //CALL BEFORE!
	}
}

__global__ void StressStrainExtKernel(double *sigma,	//OUTPUT
																								double *strain,double *straina,double *strainb, //OUTPUT
																								//INPUT
																								double *p, double *rotrate, 
																								double *shearstress,double *shearstressa, double *shearstressb,
																								
																								double dt, int particle_count) {
	// int i = threadIdx.x + blockDim.x*blockIdx.x;
		
	// if ( i < particle_count ) {	
		// //Pressure = EOS(PresEq, Cs, P0,Density, RefDensity); //CALL BEFORE!

		// // Jaumann rate terms
		// tensor3 RotationRateT,SRT,RS;
		// tensor3 RotationRate;
		
		// double temprr[6],tempss[6];
		// for (int k=0;k<6;k++){ //First the diagonal
			// temprr[k]=rotrate[6*i+k];
			// tempss[k]=shearstress[6*i+k];
		// }
		
		// RotationRate.FromFlatSym(tempss);
		// RotationRate.FromFlatAntiSym(temprr);
		// RotationRateT = temprr.Trans();
		// SRT = ;
		// Trans(RotationRate,RotationRateT);
		// Mult(ShearStress,RotationRateT,SRT);
		// Mult(RotationRate,ShearStress,RS);
		// double dep =0.;
		// double prev_sy;
		// double Et;
		
		// // Elastic prediction step (ShearStress_e n+1)
		// if (FirstStep)
			// ShearStressa	= -dt/2.0*(2.0*G*(StrainRate-1.0/3.0*(StrainRate(0,0)+StrainRate(1,1)+StrainRate(2,2))*OrthoSys::I)+SRT+RS) + ShearStress;
		// ShearStressb	= ShearStressa;
		// ShearStressa	= dt*(2.0*G*(StrainRate-1.0/3.0*(StrainRate(0,0)+StrainRate(1,1)+StrainRate(2,2))*OrthoSys::I)+SRT+RS) + ShearStressa;	

		// //Fail, TODO
		
		// ShearStress	= 1.0/2.0*(ShearStressa+ShearStressb);
		// Sigma = -p[i] * OrthoSys::I + ShearStress;	//Fraser, eq 3.32
		
		// if (FirstStep)
			// Straina	= -dt/2.0*StrainRate + Strain;
		// Strainb	= Straina;
		// Straina	= dt*StrainRate + Straina;
		// Strain	= 1.0/2.0*(Straina+Strainb);
	// }
}

__device__ void Domain_d::StressStrain(int i) {
	
	//int i = threadIdx.x + blockDim.x*blockIdx.x;
	double dep = 0.;
	
	if ( i < particle_count ) {	
		//Pressure = EOS(PresEq, Cs, P0,Density, RefDensity); //CALL BEFORE!

		// Jaumann rate terms
		tensor3 RotationRateT,SRT,RS;
		tensor3 RotationRate;
		tensor3 StrainRate;
		tensor3 ShearStress,ShearStressa,ShearStressb;
		tensor3 Sigma;
		tensor3 Strain,Straina,Strainb;
		
		double temprr[6],tempss[6],tempsr[6];
		double tempssa[6],tempssb[6];
		for (int k=0;k<6;k++){ //First the diagonal
			temprr[k] = rotrate[6*i+k];
			tempss[k] = shearstress[6*i+k];
			tempsr[k] = strrate[6*i+k];
			tempssa[k]= shearstressa[6*i+k];
			tempssb[k]= shearstressb[6*i+k];			
		}
		ShearStress   = FromFlatSym (tempss);
		ShearStressa  = FromFlatSym(tempssa);
		ShearStressb  = FromFlatSym(tempssb);
		
		StrainRate    = FromFlatSym(tempsr);
		RotationRate  = FromFlatAntiSym(temprr);

		
		RotationRateT = Trans(RotationRate);
		
		SRT = ShearStress * RotationRateT;
		RS = RotationRate * ShearStress;

		// if (i==1250){
			// printf("Stress Kernel, StrainRate\n");print(StrainRate);
			// printf("Stress Kernel, Identity() before calc\n");print((StrainRate.xx+StrainRate.yy+StrainRate.zz)* Identity());
			// printf("G, %f\n",G[i]);}
			
		// Elastic prediction step (ShearStress_e n+1)
		if (isfirst_step){
			ShearStressa	= -deltat/2.0*(2.0*G[i]*(StrainRate - 1.0/3.0*(StrainRate.xx+StrainRate.yy+StrainRate.zz)* Identity()) + SRT+RS) + ShearStress;
		}
		ShearStressb	= ShearStressa;
		ShearStressa	= deltat*(2.0*G[i]*(StrainRate-1.0/3.0*(StrainRate.xx+StrainRate.yy+StrainRate.zz)*Identity())+SRT+RS) + ShearStressa;	

		// if (i==1250){
		// printf("Stress Kernel ShearStressA\n");print(ShearStressa);}
			
		// //Fail, TODO
				
		double J2	= 0.5*(ShearStressa.xx*ShearStressa.xx + 2.0*ShearStressa.xy*ShearStressa.yx +
					2.0*ShearStressa.xz*ShearStressa.zx + ShearStressa.yy*ShearStressa.yy +
					2.0*ShearStressa.yz*ShearStressa.zy + ShearStressa.zz*ShearStressa.zz);

    //Scale back, Fraser Eqn 3-53
		double sig_trial = sqrt(3.0*J2); 
    if ( sigma_y[i] < sig_trial ) ShearStressa = sigma_y[i]/sig_trial * ShearStressa; //Yielding      
    //std::min((Sigmay/sqrt(3.0*J2)),1.0)*ShearStressa;
		
		sigma_eq[i] = sig_trial;	
		
		if ( sig_trial > sigma_y[i]) {
			dep=( sig_trial - sigma_y[i])/ (3.*G[i] /*+ Ep*/);	//Fraser, Eq 3-49 TODO: MODIFY FOR TANGENT MODULUS = 0
			pl_strain[i] += dep;	
      //printf("Particle %d, dep %.1e, sigtrial %.1e\n",i,dep,sig_trial);
			sigma_eq[i] = sigma_y[i];
		}

    
		ShearStress	= 1.0/2.0*(ShearStressa+ShearStressb);
		Sigma = -p[i] * Identity() + ShearStress;	//Fraser, eq 3.32
		// if (i == 1250){
			// printf("Time %.4e Particle 1250, pressure %f , ShearStresszz %f Sigma \n",Time, p[i], ShearStress.zz);
			// print(Sigma);
		// }
		
		if (isfirst_step)
			Straina	= -deltat/2.0*StrainRate + Strain;
		Strainb	= Straina;
		Straina	= deltat*StrainRate + Straina;
		Strain	= 1.0/2.0*(Straina+Strainb);

		///// OUTPUT TO Flatten arrays
		ToFlatSymPtr(Sigma, sigma,6*i);  //TODO: CHECK IF RETURN VALUE IS SLOWER THAN PASS AS PARAM
		
		ToFlatSymPtr(Strain, 	strain,6*i);
		ToFlatSymPtr(Straina, straina,6*i);
		ToFlatSymPtr(Strainb, strainb,6*i);
		
		ToFlatSymPtr(ShearStress, shearstress,6*i);
		ToFlatSymPtr(ShearStressa, shearstressa,6*i);
		ToFlatSymPtr(ShearStressb, shearstressb,6*i);
		
		// if (i==1250){
			// printf("Stress Strain kernel, particle 1250 Sigma\n");
			// print(Sigma);
		// }
	}//particle count
}

__global__ void StressStrainKernel(Domain_d *dom){
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	dom->StressStrain(i);
}

#define TAU		0.005
#define VMAX	10.0

// TODO #53 Make generic function pointer
// THISIS ONLY AN EXAMPLE	
__device__ void Domain_d::ApplyBCVel(int bcid, 
																		double3 bcv){
	int i = threadIdx.x + blockDim.x*blockIdx.x;	
	if ( i < particle_count ) {	
		//printf("particle %d bc \n",i);
		if (ID[i]==bcid){
			a[i]		= make_double3(0.0);
			v[i]		= bcv;
			va[i]		= bcv;
			
		}
	}
}

__global__ void ApplyBCVelKernel (Domain_d *dom, int bcid, double3 bcv) {
	
	dom->ApplyBCVel (bcid,bcv);
}

__global__ void ApplyBCVelExtKernel(	double *v, //Output
																double *va,
																int *ID, 	//Input
																int bcid, 
																double bcv,
																double Time,
																int particle_count) {
	
	int i = threadIdx.x + blockDim.x*blockIdx.x;	
	
	//VMAX/TAU * domi.getTime();
	
	if ( i < particle_count ) {	
		//if (ID[i]==bcid)
			
	}
}

__global__ void TimestepCheckKernel(const double &CFL,
																double *h,
																double *Cs){
	int i = threadIdx.x + blockDim.x*blockIdx.x;	
	
	//VMAX/TAU * domi.getTime();
	
	// if ( i < particle_count ){
		
		// t1 = CFL*h[i]/Cs[i];//Or is Cij??
	// }															

}

#include "cuda_helper.h"
#include "cuNSearch.h"

#include<array>
#include <chrono>
#include <iostream>
#include <vector>
using namespace std;
using namespace cuNSearch;
using Real3 = std::array<Real, 3>;

__global__ void testNeighboursKernel(	const uint particle,
	const uint *particlenbcount,
	const uint *neighborWriteOffsets,
	const uint *neighbors)
{
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	if (i == particle){
		printf("Particle %d nbs\n", particle);
		const uint writeOffset = neighborWriteOffsets[i];

		for (int j=0; j< particlenbcount[i];j++){
			printf("%d ", neighbors[writeOffset + j]);
		}

	}
}


void Domain_d::MechSolve(const double &tf, const double &dt_out){

	int N = particle_count;
	int threadsPerBlock = 256; //Or BlockSize
	int blocksPerGrid =				// Or gridsize
	(N + threadsPerBlock - 1) / threadsPerBlock;
  Time =0.;
	
	isfirst_step =true;

	step = 0;						//Par of domain_h
	clock_t clock_beg;
	double time_spent;
	clock_beg = clock();
	
	//TimestepCheck(0.7,h,Cs);

	double t_out;
	t_out = dt_out;
	
	double stress_time,forces_time,accel_time,pressure_time,move_time;
	
	clock_t clock_beg_int;

	stress_time = forces_time = accel_time = pressure_time = move_time = 0.;

	hipMemcpy(x_h, x, sizeof(double3) * particle_count, hipMemcpyDeviceToHost);		


	//Make the nb search at first
	
	vector < Real3> pos;
  // positions.reserve(dom.Particles.size());
	for (unsigned int i = 0; i < particle_count; i++) {
    std::array<Real, 3> x ={{ x_h[i].x,
                              x_h[i].y,
                              x_h[i].z
                            }};
		pos.push_back(x);
	}
	
	cout << "Initializing nb search data.."<<endl;
	double radius = 2.0*h_glob;
	//This bypass the original constructor 
  //TODO: make this
	
	nb_search.deviceData = std::make_unique<cuNSearchDeviceData>(radius);
	nb_search.set_radius(radius);
	
	cuNSearch::NeighborhoodSearch nsearch(radius);
	cout << "Done."<<endl;

	auto pointSetIndex = nsearch.add_point_set(pos.front().data(), pos.size(), true, true);
	auto &pointSet = nsearch.point_set(0);
	
	int *nb_part_h =  new int [particle_count * 100]; //This could be sized only once with max nb count
	int *nb_offs_h =  new int [particle_count + 1];
	
	auto points = pointSet.GetPoints();
	
	int ts_i=0;
	int ts_nb_inc = 5;
	
	bool is_yielding = false;
	double max_pl_strain = 0.;
  cout << "First Rigid Contact Particle: "<<first_fem_particle_idx<<endl;
  
	//First time find nbs
	for (int i=0; i <particle_count;i++){
	((Real3*)points)[i][0] = x_h[i].x;
	((Real3*)points)[i][1] = x_h[i].y;
	((Real3*)points)[i][2] = x_h[i].z;
	}		
	// TODO: FIX THIS! 
	//zsort is much faster than traditional, but particle order and nb changes
	//nsearch.z_sort();
	//nsearch.point_set(pointSetIndex).sort_field((Real3*)nsearch.point_set(pointSetIndex).GetPoints());
	nsearch.find_neighbors();	
		// testNeighboursKernel<<< blocksPerGrid,threadsPerBlock >>>(	0,
		// CudaHelper::GetPointer(nsearch.deviceData->d_NeighborCounts),
		// CudaHelper::GetPointer(nsearch.deviceData->d_NeighborWriteOffsets),
		// CudaHelper::GetPointer(nsearch.deviceData->d_Neighbors)
		// );
    
  int count = 1; //step
  
  //totmass = 1.;
  
  this->id_free_surf = 1;
  
  while (Time<tf) {
	
		if ( ts_i == 0 && is_yielding ){
			//cout << "Searching nbs"<<endl; 
			/////////////////////////////////////////
			// UPDATE POINTS POSITIONS
			//TODO: THIS HAS TO BE DONE WITH KERNEL
			for (int i=0; i <particle_count;i++){
			((Real3*)points)[i][0] = x_h[i].x;
			((Real3*)points)[i][1] = x_h[i].y;
			((Real3*)points)[i][2] = x_h[i].z;
			}		
			// TODO: FIX THIS! 
			//zsort is much faster than traditional, but particle order and nb changes
			//nsearch.z_sort();
			//nsearch.point_set(pointSetIndex).sort_field((Real3*)nsearch.point_set(pointSetIndex).GetPoints());
			nsearch.find_neighbors();

		}//ts_i == 0
	
		//cout << "
		
		//cout<<"--------------------------- BEGIN STEP "<<step<<" --------------------------"<<endl; 
		//This was in Original LastCompAcceleration
		clock_beg_int = clock();
		CalcForcesKernel	<<<blocksPerGrid,threadsPerBlock >>>(this,
      CudaHelper::GetPointer(nsearch.deviceData->d_NeighborCounts),
      CudaHelper::GetPointer(nsearch.deviceData->d_NeighborWriteOffsets),
      CudaHelper::GetPointer(nsearch.deviceData->d_Neighbors)		
		);
    hipDeviceSynchronize(); //REQUIRED!!!!
    
    
    if (contact){
      CalculateSurfaceKernel<<<blocksPerGrid,threadsPerBlock >>>(this,
      CudaHelper::GetPointer(nsearch.deviceData->d_NeighborCounts),
      CudaHelper::GetPointer(nsearch.deviceData->d_NeighborWriteOffsets),
      CudaHelper::GetPointer(nsearch.deviceData->d_Neighbors),		    
      /*id,*/
      totmass);
      hipDeviceSynchronize(); //REQUIRED!!!!
      //
      CalcContactNbKernel<<<blocksPerGrid,threadsPerBlock >>>(this,
      CudaHelper::GetPointer(nsearch.deviceData->d_NeighborCounts),
      CudaHelper::GetPointer(nsearch.deviceData->d_NeighborWriteOffsets),
      CudaHelper::GetPointer(nsearch.deviceData->d_Neighbors)    
      );
      hipDeviceSynchronize(); //REQUIRED!!!!    
      
      CalcContactForcesKernel<<<blocksPerGrid,threadsPerBlock >>>(this,
      CudaHelper::GetPointer(nsearch.deviceData->d_NeighborCounts),
      CudaHelper::GetPointer(nsearch.deviceData->d_NeighborWriteOffsets),
      CudaHelper::GetPointer(nsearch.deviceData->d_Neighbors) 
      );
      hipDeviceSynchronize();
    }
    
    
    //cout << "end"<<endl;
    
    forces_time += (double)(clock() - clock_beg_int) / CLOCKS_PER_SEC;
    
		//IMPOSE BC!
		ApplyBCVelKernel	<<<blocksPerGrid,threadsPerBlock >>>(this, 2, make_double3(0.,0.,0.));
		hipDeviceSynchronize();
    double vbc;
    if (Time < TAU) vbc = VMAX/TAU*Time;
    else            vbc = VMAX;
		//double vbc = 1.0; 

		ApplyBCVelKernel	<<<blocksPerGrid,threadsPerBlock >>>(this, 3, make_double3(0.,0.,-vbc));
		hipDeviceSynchronize();

		deltatmin = deltatint = deltat;
		//Save before move (to be changed)
    
    if (contact){
      MeshUpdateKernel<<<blocksPerGrid,threadsPerBlock >>>(this->trimesh, deltat);
      hipDeviceSynchronize();
    }

		
		if (Time >= t_out) {		
			hipMemcpy(ID_h, ID, sizeof(int) * particle_count, hipMemcpyDeviceToHost);	
			hipMemcpy(x_h, x, sizeof(double3) * particle_count, hipMemcpyDeviceToHost);	
			hipMemcpy(u_h, u, sizeof(double3) * particle_count, hipMemcpyDeviceToHost);	
			hipMemcpy(v_h, v, sizeof(double3) * particle_count, hipMemcpyDeviceToHost);	
			hipMemcpy(a_h, a, sizeof(double3) * particle_count, hipMemcpyDeviceToHost);	
			
			hipMemcpy(p_h, p, sizeof(double) * particle_count, hipMemcpyDeviceToHost);	
			
			hipMemcpy(rho_h, rho, sizeof(double) * particle_count, hipMemcpyDeviceToHost);
			hipMemcpy(sigma_eq_h, sigma_eq, sizeof(double) * particle_count, hipMemcpyDeviceToHost);	
			hipMemcpy(pl_strain_h, pl_strain, sizeof(double) * particle_count, hipMemcpyDeviceToHost);
      
      hipMemcpy(contneib_count_h,contneib_count, sizeof(int) * particle_count, hipMemcpyDeviceToHost);
			
			char str[10];
			sprintf(str, "out_%d.csv", count);
      count++;
			WriteCSV(str);
			
			t_out += dt_out;
			time_spent = (double)(clock() - clock_beg) / CLOCKS_PER_SEC;
			cout << "Time "<<Time<<", GPU time "<<time_spent<<endl;
			cout << "Current time step: "<< deltat << endl;
			cout << "Forces calc: "			<<forces_time<<endl;
			cout << "Stresses calc: "		<<stress_time<<endl;
			
			double3 max= make_double3(0.,0.,0.);
			for (int i=0;i<particle_count;i++){
				//cout << "Particle " << i << "Vel: "<< v_h[i].x<<", "<<v_h[i].y<< ", "<< v_h[i].z<<endl;
				//cout << "Particle " << i << "Acc: "<< a_h[i].x<<", "<<a_h[i].y<< ", "<< a_h[i].z<<endl;
				if (u_h[i].x>max.x) max.x = u_h[i].x;
				if (u_h[i].y>max.y) max.y = u_h[i].y;
				if (u_h[i].z>max.z) max.z = u_h[i].z;
			}
			cout << "Max disp "<< max.x<<", "<<max.y<<", "<<max.z<<endl;
		}
					
		//TODO: CHANGE this to an interleaved reduction or something like that (see #84)
		if (!is_yielding){
			hipMemcpy(pl_strain_h, pl_strain, sizeof(double) * particle_count, hipMemcpyDeviceToHost);
			for (int i=0;i<particle_count;i++){
				if ( pl_strain_h[i] > max_pl_strain )
					max_pl_strain = pl_strain_h[i];
			}
			
			if ( max_pl_strain > MIN_PS_FOR_NBSEARCH ){
				is_yielding = true;
				cout << "Now is yielding"<<endl;
			}
		}
	
		if (auto_ts){
			CalcMinTimeStepKernel<<< blocksPerGrid,threadsPerBlock >>> (this);
			hipDeviceSynchronize();
			
			// hipMemcpy(max_deltat_h, max_deltat, sizeof(double) * particle_count, hipMemcpyDeviceToHost);
			// double max_dt=1000.;
			// int part;
			// for (int i=0;i<particle_count;i++){
				// if ( max_deltat_h[i] < max_dt ){
					// max_dt = max_deltat_h[i];
					// deltatmin = max_dt;
					// part = i;
				// }
			// }
			//cout << "Max delta t (safe): " << max_dt<<"in particle "<<part<< ", parallel: "<<deltatmin<<endl;
			AdaptiveTimeStep();
			//cout << "Auto TS is on. Time Step size: "<<deltat<<endl;
		}

		//Move particle and then calculate streses and strains ()
		MoveKernelExt<<<blocksPerGrid,threadsPerBlock >>> (v, va,vb,
														rho, rhoa, rhob, drho,
														x, a,
														u, /*Mat3_t I, */deltat,
														isfirst_step, particle_count);	
		hipDeviceSynchronize(); //REQUIRED!!!!

		//If kernel is the external, calculate pressure
		//Calculate pressure!
		PressureKernelExt<<<blocksPerGrid,threadsPerBlock >>>(p,PresEq,Cs,P0,rho,rho_0,particle_count);
		hipDeviceSynchronize();
		// StressStrainExtKernel(sigma,	//OUTPUT
																									// double *strain,*straina,*strainb, //OUTPUT
																									// //INPUT
																									// double *p, double *rotrate, 
																									// double* shearstress,double* shearstressa, double* shearstressb,
												
																									// double dt, int particle_count);
		clock_beg_int = clock();
		StressStrainKernel<<<blocksPerGrid,threadsPerBlock >>>(this);
		hipDeviceSynchronize();
		stress_time += (double)(clock() - clock_beg_int) / CLOCKS_PER_SEC;
		
		if (isfirst_step) isfirst_step = false;
		Time +=deltat;		
		
		//TODO: Pass toPartData
		//CalcForcesMember	<<<blocksPerGrid,threadsPerBlock >>>(partdata);
		//MechSolveKernel<<< >>>();

		time_spent = (double)(clock() - clock_beg) / CLOCKS_PER_SEC;	
		step ++;
		//cout<<"--------------------------- END STEP, Time"<<Time <<", --------------------------"<<endl; 

		ts_i ++;
		if ( ts_i > (ts_nb_inc - 1) ) 
			ts_i = 0;
		
	}//while <tf


	time_spent = (double)(clock() - clock_beg) / CLOCKS_PER_SEC;
	
	printf("Total steps: %d, time spent %f\n",step, time_spent);
	
	delete nb_part_h;
	delete nb_offs_h;

}

};//SPH