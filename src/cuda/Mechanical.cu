#include "hip/hip_runtime.h"
#include "Domain_d.cuh"
namespace SPH{

//Called by __global__
//void __device__ Domain_d::WholeVelocity() {
    // //Apply a constant velocity to all particles in the initial time step
    // if (norm(BC.allv)>0.0 || BC.allDensity>0.0) {
    	// Vec3_t vel = 0.0;
    	// double den = 0.0;

    	// for (int i=0 ; i<Particles.Size() ; i++) {
		// AllCon(Particles[i]->x,vel,den,BC);
    		// if (Particles[i]->IsFree && norm(BC.allv)>0.0) {
			// Particles[i]->v		= vel;
 		// }
    		// if (IsFree[i] && BC.allDensity>0.0) {
			// rho[i]	= den;
			// p[i]		= EOS(Particles[i]->PresEq[i], Particles[i]->Cs[i], Particles[i]->P0,Particles[i]->rho[i], RefDensity);
    		// }
    	// }
    // }
//}


// __device__ PartData_d::PrimaryComputeAcceleration(){
	// // Summing the smoothed pressure, velocity and stress for fixed particles from neighbour particles
	
	
	// // Calculateing the finala value of the smoothed pressure, velocity and stress for fixed particles
// }

// void __global__ MechSolveKernel (double dt, PartData_d *partdata) {
	// int i = threadIdx.x+blockDim.x*blockIdx.x;
	// dTdt[i] = 0.;
	
	// int neibcount;
	// #ifdef FIXED_NBSIZE
	// neibcount = neib_offs[i];
	// #else
	// neibcount =	neib_offs[i+1] - neib_offs[i];
	// #endif
	// printf("Solving\n");
	// for (int k=0;k < neibcount;k++) { //Or size
		// //if fixed size i = part * NB + k
		// //int j = neib[i][k];
		// int j = NEIB(i,k);
		// printf("i,j\n",i,j);
		// double3 xij; 
		// xij = x[i] - x[j];
		// double h_ = (h[i] + h[j])/2.0;
		// double nxij = length(xij);
		
		// double GK	= GradKernel(3, 0, nxij/h_, h_);
		// //		Particles[i]->dTdt = 1./(Particles[i]->Density * Particles[i]->cp_T ) * ( temp[i] + Particles[i]->q_conv + Particles[i]->q_source);	
		// //   mc[i]=mj/dj * 4. * ( P1->k_T * P2->k_T) / (P1->k_T + P2->k_T) * ( P1->T - P2->T) * dot( xij , v )/ (norm(xij)*norm(xij));
		// dTdt[i] += m[j]/rho[j]*( 4.0*k_T[i]*k_T[j]/(k_T[i]+k_T[j]) * (T[i] - T[j])) * dot( xij , GK*xij )/(nxij*nxij);
	// }
	// dTdt[i] *=1/(rho[i]*cp[i]);
// }

__device__ __forceinline__ void Domain_d::LastComputeAcceleration(){
	//partdata->();
}

void Domain_d::MechSolve(const double &tf){

	int N = particle_count;
	int threadsPerBlock = 256; //Or BlockSize
	int blocksPerGrid =				// Or gridsize
	(N + threadsPerBlock - 1) / threadsPerBlock;
  Time =0.;
	
	isfirst_step =true;
	
	//This was in Original LastCompAcceleration
	CalcForcesKernel	<<<blocksPerGrid,threadsPerBlock >>>(this);
	hipDeviceSynchronize(); //REQUIRED!!!!
	
	//TODO: Pass toPartData
	//CalcForcesMember	<<<blocksPerGrid,threadsPerBlock >>>(partdata);
	//MechSolveKernel<<< >>>();
	
	
}

};//SPH