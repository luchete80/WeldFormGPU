#include "hip/hip_runtime.h"
#ifndef _GEOMETRY_H_
#define _GEOMETRY_H_

#include "Domain_d.cuh"
#include "Functions.cuh"
#include "tensor.cuh"
#include "tensor3.cu" //INLINE

#include <iostream>
  
namespace SPH {
  
__global__ inline void CalculateSurfaceKernel(Domain_d *dom_d,	const uint *particlenbcount,
																	const uint *neighborWriteOffsets,
																	const uint *neighbors,
																	/*const int &id, */const double &totmass){
	dom_d->CalculateSurface(
	particlenbcount,
	neighborWriteOffsets,
	neighbors,
	/*id, */dom_d->totmass);

}
                                  
// Calculate Free Surface (for contact and heat convection)
void __device__ inline Domain_d::CalculateSurface(const uint *particlenbcount,
                                                  const uint *neighborWriteOffsets,
                                                  const uint *neighbors,
                                                  /*const int &id, */const double &totmass){
	//id_free_surf = id;

	int i = threadIdx.x + blockDim.x*blockIdx.x;
	//printf ("calc surf on max id %d\n", first_fem_particle_idx[0] );
  
	if ( i < first_fem_particle_idx[0] ) { //In Contact Surface Particles, normal are updated different way
    normal[i] = make_double3(0.,0.,0.);
    int surf_part=0;
    int neibcount = particlenbcount[i];
    int nbcount_corr = 0; //WITHOUT CONTACT SURFACE!!
    const uint writeOffset = neighborWriteOffsets[i];
    
    
      for (int k=0;k < neibcount; k++) { //Or size
        //if fixed size i = part * NB + k
        //int j = neib[i][k];
        int j = neighbors[writeOffset + k];
        //double h	= partdata->h[i]+P2->h)/2;
        double3 xij = x[i] - x[j];
        bool next_to_contsurf = false;
        for (int mc=0;mc</*trimesh_count*/1;mc++)
          if (ID[j]==contact_surf_id[mc]){
            next_to_contsurf = true;
            //printf("NEXT TO CONT SURF\n");
          }
          //printf("found part %d\n",j);
          //printf("part %d ID nb%d, cont surf mc %d\n",i,ID[j],contact_surf_id[mc]);
          //if (ID[j]!=contact_surf_id[mc]){  //EXCLUDE RIGID PAIRS!
          if (!next_to_contsurf){
            normal[i] += m[j] * xij; 
            //printf("nside\n");
            // if (i==0)
            //printf("particle %d Nb %d xij: %f %f %f mj %.6e\n", i, j, xij.x, xij.y, xij.z, m[j]);
            nbcount_corr++;
          }//! next to surface
      }//neighbour
        

      
      // if (i==0)
        //printf("particle %d normal : %f %f %f , nb %d, nbc %d totmass %f\n", i, normal[i].x, normal[i].y, normal[i].z, neibcount, nbcount_corr, totmass);
      //printf("%d \n", nbcount_corr);
      normal[i]*= ((double)particle_count/(totmass *(double)nbcount_corr)); //Attention parenthesis, if not it crashes
      //normal[i]*= 1./totmass;
      // if (i==0)
        // printf("particle %d normal : %f %f %f , nb %d length %f\n", i, normal[i].x, normal[i].y, normal[i].z, nbcount_corr, length (normal[i]));
      if ( length(normal[i]) >= 0.25 * h[i] && nbcount_corr <= 46) {//3-114 Fraser {
        if (!not_write_surf_ID[i]){
          ID[i] = id_free_surf; //THIS CRASH IS ASSIGNED BY PARAMETER
          printf("ASSIGNING ID %d\n",ID[i]);
          //printf("particle %d normal : %f %f %f , nb %d\n", i, normal[i].x, normal[i].y, normal[i].z, nbcount_corr);
          //surf_part++;
        }
      }
    
    //printf("surf part %d, nbcount_corr %d not_write %d, ID %d\n",surf_part, nbcount_corr, not_write_surf_ID[i], ID[i] );
  }//i < particle_count
  
}

}; //SPH

#endif
