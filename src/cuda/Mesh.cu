#include "hip/hip_runtime.h"
// TODO: extend to all dirs
//NOTE: DENSITY IS OF ELEMENTS
//This also will be passed to device
namespace SPH{
//NOW THIS IS ZORIENTED, CHANGE TO EVERY PLANE
inline void TriMesh_d::AxisPlaneMesh(const int &axis, bool positaxisorent, const double3 p1, const double3 p2,  const int &dens){
	
	double x1,x2,x3;
	double l1,l2;
	double3 p = p2-p1;
	int dir[3];
	if 			(axis == 0 )	{dir[0] = 1; dir[1] = 2;}
	else if (axis == 1 )	{dir[0] = 0; dir[1] = 2;}
	else									{dir[0] = 0; dir[1] = 1;}
	
	dir [2] = axis; //dir2 is which remains constant
	
	x3 = p1.z;
	x2 = p1.y; 
  
  //TODO: CORRECT
  //x3 = p1.y;
  //x2 = p1.y;
  
	//double dl = p(dir[0])/dens;	//Could be allowed 2 diff densities
  double dl = p.x/dens;
  int nodecount = (dens+1)*(dens+1);
  // node = new double3 [nodecount];
  // node_v = new double3 [nodecount];
  
  //Is it necessary to paralellize mesh nodes??
  hipMalloc((void **)&node   , 	nodecount * sizeof (double3));
  hipMalloc((void **)&node_v , 	nodecount * sizeof (double3));
  
  double3 *node_h, *node_vh;
  node_h  =  new double3 [nodecount];
  node_vh =  new double3 [nodecount];
  
	//cout <<"dens: "<<dens<<endl;
	//Plane is in 0 and 1 dirs
  cout << "Creating nodes.."<<endl;
	int vi=0;
	int test =dens+1;
	for (int j=0; j<test; j++) {
		//x1 = p1(dir[0]);
    x1 = p1.x;
		for (int i=0; i<test; i++){
			double3 v;
			v.x=x1;v.y=x2;v.z=x3;
			//cout << "i,j" << i << ", " << j<<endl; 
			//node.Push(new double3(x1,x2,x3));
			node_h[vi]		=make_double3(v.x,v.y,v.z);
			node_vh[vi]	=make_double3(0.,0.,0.);
      vi++;
			// node.Push(new double3(v(0),v(1),v(2)));
			// node_v.Push(new double3(0.,0.,0.));
			//cout << "xyz: "<<x1 << ", "<<x2<<", "<<x3<<endl;
			x1+=dl;
		}
		x2+=dl;
	}
  hipMemcpy(node_h, node, nodecount, hipMemcpyHostToDevice);
  hipMemcpy(node_vh, node_v, nodecount, hipMemcpyHostToDevice);

  cout << "Element count: "<<elcount << endl;  
  cout << "done. Creating elements... ";
	int n[4];
	int el =0;
	int i;
	
	int elcount = dens * dens * 2;
	hipMalloc((void **)&centroid , 	elcount * sizeof (double3));
	hipMalloc((void **)&normal 	, 	elcount * sizeof (double3));
	hipMalloc((void **)&elnode 	, 	3 * elcount * sizeof (int));	
  int *elnode_h = new int[3*elcount];
  double3 *centroid_h = new double3[elcount];
  double3 *normal_h   = new double3[elcount];
	
	for (size_t j = 0 ;j  < dens; j++ ) {
				// cout <<"j, dens" <<j<<", "<<dens<<endl;
				// cout <<"j<dens"<< (j  < dens)<<endl;
		for ( i = 0; i < dens; i++ ){
				// cout <<"i, dens" <<i<<", "<<dens<<endl;
				// cout <<"i <dens"<< (i  < dens)<<endl;
				n[0] = (dens + 1)* j + i; 		n[1] = n[0] + 1; 
				n[2] = (dens + 1)* (j+1) + i; n[3] = n[2] + 1;
			//cout <<" jj" << jj<<endl;
			int elcon[2][3];	// TODO: check x, y and z normals and node direction 
												// For all plane orientations
			//If connectivity  is anticlockwise normal is outwards
			if (positaxisorent) {
				elcon[0][0] = n[0];elcon[0][1] = n[1];elcon[0][2] = n[2];
				elcon[1][0] = n[1];elcon[1][1] = n[3];elcon[1][2] = n[2];
			} else {
				elcon[0][0] = n[0];elcon[0][1] = n[2];elcon[0][2] = n[1];
				elcon[1][0] = n[1];elcon[1][1] = n[2];elcon[1][2] = n[3];				
			}
			for ( int e= 0; e<2;e++) { // 2 triangles
				int elnodeid = 3*el;
				//element.Push(new Element(elcon[e][0],elcon[e][1],elcon[e][2]));		
				elnode_h[elnodeid + 0] = elcon[e][0]; 
				elnode_h[elnodeid + 1] = elcon[e][1]; 
				elnode_h[elnodeid + 2] = elcon[e][2];
				//cout << "Element "<< el <<": ";
				// for (int en = 0 ; en<3; en++) cout << elcon[e][en]<<", ";
				// cout <<endl;
				
				double3 v = ( node_h[elcon[e][0]] + node_h[elcon[e][1]] + node_h[elcon[e][2]] ) / 3. ;
				//element[el] -> centroid = v; 
				centroid_h[el] = v;
				//cout << "Centroid" << element[el] -> centroid << endl;
				el++;
			}
		}// i for
		
	}

	///////////////////////////////////////////
	//// MESH GENERATION END
	cout << endl<<"Done. Creating normals"<<endl;
	for (int e = 0; e < elcount; e++){ 
		double f=-1.;
		if (positaxisorent) f= 1.;
		//element[e] -> normal (axis) = f;
		if (axis == 0)			normal_h[e].x = f;
		else if (axis == 1)	normal_h[e].y = f;
		else 								normal_h[e].z = f;
	}

  hipMemcpy(elnode_h, elnode, elcount, hipMemcpyHostToDevice);
  hipMemcpy(centroid_h, centroid, elcount, hipMemcpyHostToDevice);
  hipMemcpy(normal_h, normal, elcount, hipMemcpyHostToDevice);

  delete node_h;
  delete elnode_h;
  delete centroid_h;
  delete normal_h;  
}

inline __device__ void TriMesh_d::UpdatePlaneCoeff(){
	//Update pplan
	//	pplane[i] = dot(*node [element[e] -> node[element[e] ->nfar]],element[e] -> normal);

}

};