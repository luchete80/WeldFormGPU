#include "hip/hip_runtime.h"
// TODO: extend to all dirs
//NOTE: DENSITY IS OF ELEMENTS
//This also will be passed to device
namespace SPH{
//NOW THIS IS ZORIENTED, CHANGE TO EVERY PLANE
inline void TriMesh_d::AxisPlaneMesh(const int &axis, bool positaxisorent, const double3 p1, const double3 p2,  const int &dens){
	
	double x1,x2,x3;
	double l1,l2;
	double3 p = p2-p1;
	int dir[3];
	if 			(axis == 0 )	{dir[0] = 1; dir[1] = 2;}
	else if (axis == 1 )	{dir[0] = 0; dir[1] = 2;}
	else									{dir[0] = 0; dir[1] = 1;}
	
	dir [2] = axis; //dir2 is which remains constant
	
	x3 = p1.z;
	x2 = p1.y; 
  
  //TODO: CORRECT
  //x3 = p1.y;
  //x2 = p1.y;
  
	//double dl = p(dir[0])/dens;	//Could be allowed 2 diff densities
  double dl = p.x/dens;
  int nodecount = (dens+1)*(dens+1);
  // node = new double3 [nodecount];
  // node_v = new double3 [nodecount];
  
  //Is it necessary to paralellize mesh nodes??
  hipMalloc((void **)&node   , 	nodecount * sizeof (double3));
  hipMalloc((void **)&node_v , 	nodecount * sizeof (double3));
  
  double3 *node_h, *node_vh;
  node_h  =  new double3 [nodecount];
  node_vh =  new double3 [nodecount];
  
	//cout <<"dens: "<<dens<<endl;
	//Plane is in 0 and 1 dirs
  cout << "Creating nodes.."<<endl;
	int vi=0;
	int test =dens+1;
	for (int j=0; j<test; j++) {
		//x1 = p1(dir[0]);
    x1 = p1.x;
		for (int i=0; i<test; i++){
			double3 v;
			v.x=x1;v.y=x2;v.z=x3;
			//cout << "i,j" << i << ", " << j<<endl; 
			//node.Push(new double3(x1,x2,x3));
			node_h[vi]		=make_double3(v.x,v.y,v.z);
			node_vh[vi]	=make_double3(0.,0.,0.);
      vi++;
			// node.Push(new double3(v(0),v(1),v(2)));
			// node_v.Push(new double3(0.,0.,0.));
			//cout << "xyz: "<<x1 << ", "<<x2<<", "<<x3<<endl;
			x1+=dl;
		}
		x2+=dl;
	}
  hipMemcpy(node_h, node, nodecount, hipMemcpyHostToDevice);
  hipMemcpy(node_vh, node_v, nodecount, hipMemcpyHostToDevice);

  cout << "Element count: "<<elcount << endl;  
  cout << "done. Creating elements... ";
	int n[4];
	int el =0;
	int i;
	
	int elcount = dens * dens * 2;
	hipMalloc((void **)&centroid , 	elcount * sizeof (double3));
	hipMalloc((void **)&normal 	, 	elcount * sizeof (double3));
	hipMalloc((void **)&elnode 	, 	3 * elcount * sizeof (int));	
  int *elnode_h = new int[3*elcount];
  double3 *centroid_h = new double3[elcount];
  double3 *normal_h   = new double3[elcount];
	
	for (size_t j = 0 ;j  < dens; j++ ) {
				// cout <<"j, dens" <<j<<", "<<dens<<endl;
				// cout <<"j<dens"<< (j  < dens)<<endl;
		for ( i = 0; i < dens; i++ ){
				// cout <<"i, dens" <<i<<", "<<dens<<endl;
				// cout <<"i <dens"<< (i  < dens)<<endl;
				n[0] = (dens + 1)* j + i; 		n[1] = n[0] + 1; 
				n[2] = (dens + 1)* (j+1) + i; n[3] = n[2] + 1;
			//cout <<" jj" << jj<<endl;
			int elcon[2][3];	// TODO: check x, y and z normals and node direction 
												// For all plane orientations
			//If connectivity  is anticlockwise normal is outwards
			if (positaxisorent) {
				elcon[0][0] = n[0];elcon[0][1] = n[1];elcon[0][2] = n[2];
				elcon[1][0] = n[1];elcon[1][1] = n[3];elcon[1][2] = n[2];
			} else {
				elcon[0][0] = n[0];elcon[0][1] = n[2];elcon[0][2] = n[1];
				elcon[1][0] = n[1];elcon[1][1] = n[2];elcon[1][2] = n[3];				
			}
			for ( int e= 0; e<2;e++) { // 2 triangles
				int elnodeid = 3*el;
				//element.Push(new Element(elcon[e][0],elcon[e][1],elcon[e][2]));		
				elnode_h[elnodeid + 0] = elcon[e][0]; 
				elnode_h[elnodeid + 1] = elcon[e][1]; 
				elnode_h[elnodeid + 2] = elcon[e][2];
				//cout << "Element "<< el <<": ";
				// for (int en = 0 ; en<3; en++) cout << elcon[e][en]<<", ";
				// cout <<endl;
				
				double3 v = ( node_h[elcon[e][0]] + node_h[elcon[e][1]] + node_h[elcon[e][2]] ) / 3. ;
				//element[el] -> centroid = v; 
				centroid_h[el] = v;
				//cout << "Centroid" << element[el] -> centroid << endl;
				el++;
			}
		}// i for
		
	}

	///////////////////////////////////////////
	//// MESH GENERATION END
	cout << endl<<"Done. Creating normals"<<endl;
	for (int e = 0; e < elcount; e++){ 
		double f=-1.;
		if (positaxisorent) f= 1.;
		//element[e] -> normal (axis) = f;
		if (axis == 0)			normal_h[e].x = f;
		else if (axis == 1)	normal_h[e].y = f;
		else 								normal_h[e].z = f;
	}
  
  hipMalloc((void **)&pplane , 	elcount * sizeof (double));
  hipMalloc((void **)&nfar   , 	elcount * sizeof (int));
  
  hipMemcpy(elnode_h, elnode, elcount, hipMemcpyHostToDevice);
  hipMemcpy(centroid_h, centroid, elcount, hipMemcpyHostToDevice);
  hipMemcpy(normal_h, normal, elcount, hipMemcpyHostToDevice);

  delete node_h;
  delete elnode_h;
  delete centroid_h;
  delete normal_h;  
}

//This is done once, Since mesh is rigid
//Calculate radius and plane coefficient
inline __device__ void TriMesh_d::CalcSpheres(){
	// double max;
  int e = threadIdx.x + blockDim.x*blockIdx.x;
  double max = 0.;
  double3 rv;
  for (int n = 0 ;n < 3; n++){
    rv = node[3*e+n] - centroid[e];
    if (length(rv) > max) max = length(rv);
    nfar[e] = n;
  }
	
  //element[e]-> radius[e] = max;	//Fraser Eq 3-136
	
	UpdatePlaneCoeff();
	
}

inline __device__ void TriMesh_d::UpdatePlaneCoeff(){
	//Update pplan
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  if (i < elcount) { //parallelize by element
    pplane[i] = dot(node[elnode[nfar[i]]],normal[i]);
  }
}

inline __device__ void TriMesh_d::CalcNormals(){
	double3 u, v, w;
  int e = threadIdx.x + blockDim.x*blockIdx.x;

  u = node [elnode[3*e+1]] - node [elnode[3*e]];
  v = node [elnode[3*e+2]] - node [elnode[3*e]];
  w = cross(u,v);
  normal[e] = w/length(w);
  //Fraser Eqn 3.34
  //Uj x Vj / |UjxVj|
	
}

};