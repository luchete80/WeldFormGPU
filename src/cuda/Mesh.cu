#include "hip/hip_runtime.h"
// TODO: extend to all dirs
//NOTE: DENSITY IS OF ELEMENTS
inline void TriMesh::AxisPlaneMesh(const int &axis, bool positaxisorent, const double3 p1, const double3 p2,  const int &dens){
	int elemcount = dens * dens;
	
	double x1,x2,x3;
	double l1,l2;
	double3 p = p2-p1;
	int dir[3];
	if 			(axis == 0 )	{dir[0] = 1; dir[1] = 2;}
	else if (axis == 1 )	{dir[0] = 0; dir[1] = 2;}
	else									{dir[0] = 0; dir[1] = 1;}
	
	dir [2] = axis; //dir2 is which remains constant
	
	x3 = p1(dir[2]);

	x2=p1(dir[1]); 
	double dl = p(dir[0])/dens;	//Could be allowed 2 diff densities
  
  int nodecount = (dens+1)*(dens+1);
  // node = new double3 [nodecount];
  // node_v = new double3 [nodecount];
  
  //Is it necessary to paralellize mesh nodes??
  hipMalloc((void **)&node   , 	nodecount * sizeof (double3));
  hipMalloc((void **)&node_v , 	nodecount * sizeof (double3));
  
	//cout <<"dens: "<<dens<<endl;
	//Plane is in 0 and 1 dirs
	int v=0;
	int test =dens+1;
	for (int j=0; j<test; j++) {
		x1 = p1(dir[0]);
		for (int i=0; i<test; i++){
			double3 v;
			v(dir[0])=x1;v(dir[1])=x2;v(dir[2])=x3;
			//cout << "i,j" << i << ", " << j<<endl; 
			//node.Push(new double3(x1,x2,x3));
			node[v]		=make_double3(v(0),v(1),v(2));
			node_v[v]	=make_double3(0.,0.,0.);
			// node.Push(new double3(v(0),v(1),v(2)));
			// node_v.Push(new double3(0.,0.,0.));
			//cout << "xyz: "<<x1 << ", "<<x2<<", "<<x3<<endl;
			x1+=dl;
		}
		x2+=dl;
	}

	int n[4];
	int el =0;
	int i;
	
	int elcount = dens * dens * 2;
	hipMalloc((void **)&elem_data.centroid , 	elcount * sizeof (double3));
	hipMalloc((void **)&elem_data.normal 	, 	elcount * sizeof (double3));
	hipMalloc((void **)&elem_data.node 		, 	3 * elcount * sizeof (int));	
	
	for (size_t j = 0 ;j  < dens; j++ ) {
				// cout <<"j, dens" <<j<<", "<<dens<<endl;
				// cout <<"j<dens"<< (j  < dens)<<endl;
		for ( i = 0; i < dens; i++ ){
				// cout <<"i, dens" <<i<<", "<<dens<<endl;
				// cout <<"i <dens"<< (i  < dens)<<endl;
				n[0] = (dens + 1)* j + i; 		n[1] = n[0] + 1; 
				n[2] = (dens + 1)* (j+1) + i; n[3] = n[2] + 1;
			//cout <<" jj" << jj<<endl;
			int elcon[2][3];	// TODO: check x, y and z normals and node direction 
												// For all plane orientations
			//If connectivity  is anticlockwise normal is outwards
			if (positaxisorent) {
				elcon[0][0] = n[0];elcon[0][1] = n[1];elcon[0][2] = n[2];
				elcon[1][0] = n[1];elcon[1][1] = n[3];elcon[1][2] = n[2];
			} else {
				elcon[0][0] = n[0];elcon[0][1] = n[2];elcon[0][2] = n[1];
				elcon[1][0] = n[1];elcon[1][1] = n[2];elcon[1][2] = n[3];				
			}
			//cout << "elnodes"<<endl;
			for ( int e= 0; e<2;e++) { // 2 triangles
				int elnodeid = 3*el;
				//element.Push(new Element(elcon[e][0],elcon[e][1],elcon[e][2]));		
				elem_data.node[elnodeid + 0] = elcon[e][0]; 
				elem_data.node[elnodeid + 1] = elcon[e][1]; 
				elem_data.node[elnodeid + 2] = elcon[e][2];
				//cout << "Element "<< el <<": ";
				// for (int en = 0 ; en<3; en++) cout << elcon[e][en]<<", ";
				// cout <<endl;
				
				double3 v = ( *node[elcon[e][0]] + *node[elcon[e][1]] + *node[elcon[e][2]] ) / 3. ;
				//element[el] -> centroid = v; 
				elem_data.centroid[el] = v;
				//cout << "Centroid" << element[el] -> centroid << endl;
				el++;
			}
		}// i for
		
	}
	///////////////////////////////////////////
	//// MESH GENERATION END
	cout << "Creating normals"<<endl;
	for (int e = 0; e < element.Size(); e++){ 
		double f=-1.;
		if (positaxisorent) f= 1.;
		//element[e] -> normal (axis) = f;
		if (axis == 0)			elem_data.normal[e].x = f;
		else if (axis == 1)	elem_data.normal[e].y = f;
		else 								elem_data.normal[e].z = f;
	}

}
