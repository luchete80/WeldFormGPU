#include "hip/hip_runtime.h"
/***********************************************************************************
* PersianSPH - A C++ library to simulate Mechanical Systems (solids, fluids        * 
*             and soils) using Smoothed Particle Hydrodynamics method              *   
* Copyright (C) 2013 Maziar Gholami Korzani and Sergio Galindo-Torres              *
*                                                                                  *
* This file is part of PersianSPH                                                  *
*                                                                                  *
* This is free software; you can redistribute it and/or modify it under the        *
* terms of the GNU General Public License as published by the Free Software        *
* Foundation; either version 3 of the License, or (at your option) any later       *
* version.                                                                         *
*                                                                                  *
* This program is distributed in the hope that it will be useful, but WITHOUT ANY  *
* WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A  *
* PARTICULAR PURPOSE. See the GNU General Public License for more details.         *
*                                                                                  *
* You should have received a copy of the GNU General Public License along with     *
* PersianSPH; if not, see <http://www.gnu.org/licenses/>                           *
************************************************************************************/

#include "Domain.h"

#include "cuda/Domain_d.cuh" 

#define TAU		0.005
#define VMAX	1.0

//#include "Vector.h"


void UserAcc(SPH::Domain & domi)
{
	// double vtraction;

	// if (domi.getTime() < TAU ) 
		// vtraction = VMAX/TAU * domi.getTime();
	// else
		// vtraction = VMAX;
	
	// #pragma omp parallel for schedule (static) num_threads(domi.Nproc)

	// #ifdef __GNUC__
	// for (size_t i=0; i<domi.Particles.size(); i++)
	// #else
	// for (int i=0; i<domi.Particles.size(); i++)
	// #endif
	
	// {
		// if (domi.Particles[i]->ID == 3)
		// {
			// domi.Particles[i]->a		= Vector(0.0,0.0,0.0);
			// domi.Particles[i]->v		= Vector(0.0,0.0,vtraction);
			// domi.Particles[i]->va		= Vector(0.0,0.0,vtraction);
			// domi.Particles[i]->vb		= Vector(0.0,0.0,vtraction);
// //			domi.Particles[i]->VXSPH	= Vector(0.0,0.0,0.0);
		// }
		// if (domi.Particles[i]->ID == 2)
		// {
			// domi.Particles[i]->a		= Vector(0.0,0.0,0.0);
			// domi.Particles[i]->v		= Vector(0.0,0.0,0.0);
			// domi.Particles[i]->vb		= Vector(0.0,0.0,0.0);
			// domi.Particles[i]->VXSPH	= Vector(0.0,0.0,0.0);
		// }
	// }
}

void report_gpu_mem()
{
    size_t free, total;
    hipMemGetInfo(&free, &total);
    std::cout << "Free = " << free << " Total = " << total <<std::endl;
}


using std::cout;
using std::endl;

//__host__		SPH::Domain dom;
	
int main(int argc, char **argv) //try
{
	cout << "Initializing"<<endl;
	SPH::Domain dom;//Cannot be defined as _device
	// //OR cudamalloc((void**)&correctBool, sizeof(int));
	// hipMallocManaged(&dom, sizeof(SPH::Domain));
	// new(dom) SPH::Domain();
	
	SPH::Domain_d *dom_d;
	report_gpu_mem();
	hipMallocManaged(&dom_d, sizeof(SPH::Domain));
	report_gpu_mem();
	dom_d->SetDimension(dom.Particles.size());

  dom.Dimension	= 3;
  dom.Nproc	= 4;
  //dom.Kernel_Set(Qubic_Spline);

//  dom.Scheme	= 0;
//     	dom.XSPH	= 0.5; //Very important

	double dx,h,rho,K,G,Cs,Fy;
	double H,L,n;

	H	= 1.;
	n	= 15.0;

	rho	= 1000.0;
	dx	= H / n;
	h	= dx*1.2; //Very important
	Cs	= sqrt(K/rho);

  double timestep;

	// cout<<"t  = "<<timestep<<endl;
	// cout<<"Cs = "<<Cs<<endl;
	// cout<<"K  = "<<K<<endl;
	// cout<<"G  = "<<G<<endl;
	// cout<<"Fy = "<<Fy<<endl;
	
	// dom.GeneralAfter = & UserAcc;
	// dom.DomMax(0) = H;
	// dom.DomMin(0) = -H;
	cout << "Creating Domain"<<endl;
	dom.AddBoxLength(1 ,Vector ( -H/2.0 -H/20., -H/2.0 -H/20., -H/2.0 -H/20. ), H + H/20., H +H/20.,  H + H/20. , dx/2.0 ,rho, h, 1 , 0 , false, false );

  //SPH::Domain	dom;
	double3 *x =  (double3 *)malloc(dom.Particles.size());
	for (int i=0;i<dom.Particles.size();i++){
		x[i] = make_double3(dom.Particles[i]->x);
	}
	int size = dom.Particles.size() * sizeof(double3);
	cout << "Copying to device..."<<endl;
	hipMemcpy(dom_d->x, x, size, hipMemcpyHostToDevice);
	cout << "copied"<<endl;
	
	// //Temporary, NB Search in GPU
	cout << "Cell Initiate..."<<endl; dom.CellInitiate();
	// cout << "Generating List..."<<endl;	dom.ListGenerate();
	
	// cout << "Nb Searching..."<<endl;	dom.MainNeighbourSearch(); 
	// cout << "Done"<<endl;
	
	// std::vector <int> nb(dom.Particles.size());
	// //std::vector <int> nbcount(Particles.size());
	
	// #pragma omp parallel for schedule (static) num_threads(Nproc)
	// for ( int k = 0; k < dom.Nproc ; k++) {
		// for (int a=0; a<dom.SMPairs[k].size();a++) {//Same Material Pairs, Similar to Domain::LastComputeAcceleration ()
			// nb[dom.SMPairs[k][a].first ]+=1;
			// nb[dom.SMPairs[k][a].second]+=1;
		// }
	// }	
	
	// cout << "Nb count"<< nb[0]<<endl;
		
		// // std::cout << "Particle Number: "<< dom.Particles.size() << endl;
     	// // double x;

    	// // for (size_t a=0; a<dom.Particles.Size(); a++)
    	// // {
    		// // x = dom.Particles[a]->x(0);
			// // dom.Particles[a]->k_T			=	3000.;
			// // dom.Particles[a]->cp_T			=	1.;
			// // dom.Particles[a]->h_conv		= 100.0; //W/m2-K
			// // dom.Particles[a]->T_inf 		= 500.;
			// // dom.Particles[a]->T				= 20.0;			
    		// // if ( x < -H/2.0 ) {
    			// // dom.Particles[a]->ID 			= 2;
    			// // dom.Particles[a]->Thermal_BC 	= TH_BC_CONVECTION;
				// // // cout << "Particle " << a << "is convection BC" <<endl;
			// // }
    	// // }

        // // timestep = (0.3*h*h*rho*dom.Particles[0]->cp_T/dom.Particles[0]->k_T);	
		// // cout << "Time Step: "<<timestep<<endl;
		// // //timestep=1.e-6;
		// // //0.3 rho cp h^2/k
	
		
	// dom.WriteCSV("maz");
	
	// WriteCSV_kernel<<<1,1>>>(&dom);
// // //    	dom.Solve(/*tf*/0.01,/*dt*/timestep,/*dtOut*/0.001,"test06",999);

		// dom.Solve(/*tf*/1.01,/*dt*/timestep,/*dtOut*/0.1,"test06",999);
		
        // return 0;
				
	hipFree(dom_d);
	report_gpu_mem();
}
//MECHSYS_CATCH
